#include "hip/hip_runtime.h"
#include "stdio.h"
#include "BlendColorMap.h"
#include "device_utils.h"
#include <hip/hip_vector_types.h>
#include ""

__global__ void BlendColorMapKernal(PtrStepSz<uchar4> rgba1, PtrStepSz<uchar4> rgba2, PtrStepSz<uchar4> rgbaDst)
{
	int u = threadIdx.x + blockIdx.x * blockDim.x;
	int v = threadIdx.y + blockIdx.y * blockDim.y;

	if (u < rgba1.cols && v < rgba1.rows)
	{
		uchar4 color;
		int a1, a2, a3;
		a1 = rgba1.ptr(v)[u].w;
		a2 = rgba2.ptr(v)[u].w;
		if (a1 == 0 && a2 == 0)
		{
			color.x = color.y = color.z = color.w = 1;
			return;
		}
		//if(rgba1.ptr(v)[u].x == 0 && rgba1.ptr(v)[u].y == 0 && rgba1.ptr(v)[u].z == 0)
		//	a1 = 0;
		a1 *= 6;
		color.x = (rgba1.ptr(v)[u].x * a1 + rgba2.ptr(v)[u].x * a2)/(a1+a2);
		color.y = (rgba1.ptr(v)[u].y * a1 + rgba2.ptr(v)[u].y * a2)/(a1+a2);
		color.z = (rgba1.ptr(v)[u].z * a1 + rgba2.ptr(v)[u].z * a2)/(a1+a2);
		rgbaDst.ptr(v)[u] = color;
	}
}
//__global__ __forceinline__ void blendColorMapKernel
void BlendColorMap(const dfusion::ColorMap & map1, const dfusion::ColorMap & map2, dfusion::ColorMap &dst)
{
	if (map1.cols() != map2.cols() || map2.rows() != map2.rows())
	{
		printf("BlendColorMap: size mismatch\n");
		return;
	}
	dst.release();
	dst.create(map1.rows(), map1.cols());

	dim3 block(32, 8);
	dim3 grid(1, 1, 1);
	grid.x = divUp(map1.cols(), block.x);
	grid.y = divUp(map1.rows(), block.y);
	BlendColorMapKernal << <grid, block >> > (map1, map2, dst);
	cudaSafeCall(hipGetLastError(), "BlendColorMap");
}
#include "hip/hip_runtime.h"
#include "GpuGaussNewtonSolver.h"
#include "device_utils.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\ModerGpuWrapper.h"
#include <iostream>
#include "GpuCholeSky.h"
namespace dfusion
{
//#define DEFINE_USE_HALF_GRAPH_EDGE
//#define CALC_DATA_TERM_NUMERIC
//#define CALC_REG_TERM_NUMERIC
//#define DEBUG_ASSIGN_10M_TO_NO_CORR
//#define DEBUG_ASSIGN_BIG_ENERGY_TO_NO_CORR
//#define ENABLE_ANTI_PODALITY

#ifdef DEFINE_USE_HALF_GRAPH_EDGE
	enum{RowPerNode_RegTerm = 3};
#else
	enum{ RowPerNode_RegTerm = 6 };
#endif
//#define USE_L2_NORM_DATA_TERM
//#define USE_L2_NORM_REG_TERM
#define CHECK(a, msg){if(!(a)) throw std::exception(msg);} 
#define CHECK_LE(a, b){if((a) > (b)) {std::cout << "" << #a << "(" << a << ")<=" << #b << "(" << b << ")";throw std::exception(" ###error!");}} 

	texture<KnnIdx, hipTextureType1D, hipReadModeElementType> g_nodesKnnTex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_nodesVwTex;
	texture<float, hipTextureType1D, hipReadModeElementType> g_twistTex;

	__device__ __forceinline__ float4 get_nodesVw(int i)
	{
		return tex1Dfetch(g_nodesVwTex, i);
	}

	__device__ __forceinline__ KnnIdx get_nodesKnn(int i)
	{
		return tex1Dfetch(g_nodesKnnTex, i);
	}

	__device__ __forceinline__ void get_twist(int i, Tbx::Vec3& r, Tbx::Vec3& t)
	{
		int i6 = i * 6;
		r.x = tex1Dfetch(g_twistTex, i6++);
		r.y = tex1Dfetch(g_twistTex, i6++);
		r.z = tex1Dfetch(g_twistTex, i6++);
		t.x = tex1Dfetch(g_twistTex, i6++);
		t.y = tex1Dfetch(g_twistTex, i6++);
		t.z = tex1Dfetch(g_twistTex, i6++);
	}

	__device__ __forceinline__ float3 read_float3_4(float4 a)
	{
		return make_float3(a.x, a.y, a.z);
	}

	__device__ __forceinline__ float sqr(float a)
	{
		return a*a;
	}

	__device__ __forceinline__ float pow3(float a)
	{
		return a*a*a;
	}

	__device__ __forceinline__ float sign(float a)
	{
		return (a>0.f) - (a<0.f);
	}

	__device__ __forceinline__ void sort_knn(KnnIdx& knn)
	{
		for (int i = 1; i < KnnK; i++)
		{
			KnnIdxType x = knn_k(knn,i);
			int	j = i;
			while (j > 0 && knn_k(knn, j - 1) > x)
			{
				knn_k(knn, j) = knn_k(knn, j - 1);
				j = j - 1;
			}
			knn_k(knn, j) = x;
		}
	}

#pragma region --bind textures
	void GpuGaussNewtonSolver::bindTextures()
	{
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<KnnIdx>();
			hipBindTexture(&offset, &g_nodesKnnTex, m_nodesKnn.ptr(), &desc,
				m_nodesKnn.size() * sizeof(KnnIdx));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error1!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
			hipBindTexture(&offset, &g_nodesVwTex, m_nodesVw.ptr(), &desc,
				m_nodesVw.size() * sizeof(float4));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error2!");
		}
		if (1)
		{
			size_t offset;
			hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			hipBindTexture(&offset, &g_twistTex, m_twist.ptr(), &desc,
				m_twist.size() * sizeof(float));
			if (offset != 0)
				throw std::exception("GpuGaussNewtonSolver::bindTextures(): non-zero-offset error3!");
		}
	}

	void GpuGaussNewtonSolver::unBindTextures()
	{
		hipUnbindTexture(g_twistTex);
		hipUnbindTexture(g_nodesVwTex);
		hipUnbindTexture(g_nodesKnnTex);
	}
#pragma endregion

#pragma region --calc data term
	struct DataTermCombined
	{
		enum
		{
			CTA_SIZE_X = GpuGaussNewtonSolver::CTA_SIZE_X,
			CTA_SIZE_Y = GpuGaussNewtonSolver::CTA_SIZE_Y,
			CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			LowerPartNum = GpuGaussNewtonSolver::LowerPartNum,
		};

		PtrStep<float4> vmap_live;
		PtrStep<float4> nmap_live;
		PtrStep<float4> vmap_warp;
		PtrStep<float4> nmap_warp;
		PtrStep<float4> vmap_cano;
		PtrStep<float4> nmap_cano;
		PtrStep<KnnIdx> vmapKnn;
		float* Hd_;
		float* g_;

		Intr intr;
		Tbx::Transfo Tlw_inv;
		Tbx::Transfo Tlw;

		int imgWidth;
		int imgHeight;
		int nNodes;

		float distThres;
		float angleThres;
		float psi_data;

		float* totalEnergy;

		__device__ __forceinline__ float data_term_energy(float f)const
		{
#ifdef USE_L2_NORM_DATA_TERM
			return 0.5f*f*f;
#else
			// the robust Tukey penelty gradient
			if (abs(f) <= psi_data)
				return psi_data*psi_data / 6.f *(1 - pow(1 - sqr(f / psi_data), 3));
			else
				return psi_data*psi_data / 6.f;
#endif
		}

		__device__ __forceinline__ float data_term_penalty(float f)const
		{
#ifdef USE_L2_NORM_DATA_TERM
			return f;
#else
			return f * sqr(max(0.f, 1.f - sqr(f / psi_data)));
			//// the robust Tukey penelty gradient
			//if (abs(f) <= psi_data)
			//	return f * sqr(1 - sqr(f / psi_data));
			//else
			//	return 0;
#endif
		}

		__device__ __forceinline__ float trace_AtB(Tbx::Transfo A, Tbx::Transfo B)const
		{
			float sum = 0;
			for (int i = 0; i < 16; i++)
				sum += A[i] * B[i];
			return sum;
		}

		__device__ __forceinline__ Tbx::Transfo compute_p_f_p_T(const Tbx::Vec3& n,
			const Tbx::Point3& v, const Tbx::Point3& vl, const Tbx::Dual_quat_cu& dq)const
		{
			//Tbx::Transfo T = Tlw*dq.to_transformation_after_normalize();
			//Tbx::Transfo nvt = outer_product(n, v);
			//Tbx::Transfo vlnt = outer_product(n, vl).transpose();
			//Tbx::Transfo p_f_p_T = T*(nvt + nvt.transpose()) - vlnt;
			Tbx::Vec3 Tn = dq.rotate(n);
			Tbx::Point3 Tv(dq.transform(v) - vl);
			return Tbx::Transfo(
				Tn.x*v.x + n.x*Tv.x, Tn.x*v.y + n.y*Tv.x, Tn.x*v.z + n.z*Tv.x, Tn.x,
				Tn.y*v.x + n.x*Tv.y, Tn.y*v.y + n.y*Tv.y, Tn.y*v.z + n.z*Tv.y, Tn.y,
				Tn.z*v.x + n.x*Tv.z, Tn.z*v.y + n.y*Tv.z, Tn.z*v.z + n.z*Tv.z, Tn.z,
				n.x, n.y, n.z, 0
				);
		}

		__device__ __forceinline__ Tbx::Transfo p_T_p_alphak_func(const Tbx::Dual_quat_cu& p_qk_p_alpha,
			const Tbx::Dual_quat_cu& dq_bar, const Tbx::Dual_quat_cu& dq, float inv_norm_dq_bar, float wk_k)const
		{
			Tbx::Transfo p_T_p_alphak = Tbx::Transfo::empty();

			float pdot = dq_bar.get_non_dual_part().dot(p_qk_p_alpha.get_non_dual_part())
				* sqr(inv_norm_dq_bar);

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[0] - dq_bar[0] * pdot
				);
			p_T_p_alphak[1] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[6] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[1] - dq_bar[1] * pdot
				);
			p_T_p_alphak[1] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[1] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[2] - dq_bar[2] * pdot
				);
			p_T_p_alphak[0] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[3] += dq[7] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[6] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[4] * p_dqi_p_alphak;
			p_T_p_alphak[8] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[10] += -dq[2] * p_dqi_p_alphak * 2;
			p_T_p_alphak[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[3] - dq_bar[3] * pdot
				);
			p_T_p_alphak[0] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[1] += -dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[2] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[3] += -dq[6] * p_dqi_p_alphak;
			p_T_p_alphak[4] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[5] += -dq[3] * p_dqi_p_alphak * 2;
			p_T_p_alphak[6] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[5] * p_dqi_p_alphak;
			p_T_p_alphak[8] += dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[9] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[4] - dq_bar[4] * pdot
				);
			p_T_p_alphak[3] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[5] - dq_bar[5] * pdot
				);
			p_T_p_alphak[3] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[6] - dq_bar[6] * pdot
				);
			p_T_p_alphak[3] += -dq[3] * p_dqi_p_alphak;
			p_T_p_alphak[7] += dq[0] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = wk_k * (
				p_qk_p_alpha[7] - dq_bar[7] * pdot
				);
			p_T_p_alphak[3] += dq[2] * p_dqi_p_alphak;
			p_T_p_alphak[7] += -dq[1] * p_dqi_p_alphak;
			p_T_p_alphak[11] += dq[0] * p_dqi_p_alphak;

			return p_T_p_alphak;
		}

		__device__ __forceinline__ bool search(int x, int y, Tbx::Point3& vl) const
		{
			float3 vwarp = read_float3_4(vmap_warp(y, x));
			float3 nwarp = read_float3_4(nmap_warp(y, x));

			return search(vwarp, nwarp, vl);
		}

		__device__ __forceinline__ bool search(float3 vwarp, float3 nwarp, Tbx::Point3& vl) const
		{
			if (isnan(nwarp.x) || isnan(vwarp.x))
				return false;

			float3 uvd = intr.xyz2uvd(vwarp);
			int2 ukr = make_int2(__float2int_rn(uvd.x), __float2int_rn(uvd.y));

			// we use opengl coordinate, thus world.z should < 0
			if (ukr.x < 0 || ukr.y < 0 || ukr.x >= imgWidth || ukr.y >= imgHeight || vwarp.z >= 0)
				return false;

			float3 vlive = read_float3_4(vmap_live[ukr.y*imgWidth + ukr.x]);
			float3 nlive = read_float3_4(nmap_live[ukr.y*imgWidth + ukr.x]);
			if (isnan(nlive.x) || isnan(vlive.x))
				return false;

#ifndef DEBUG_ASSIGN_10M_TO_NO_CORR
			float dist = norm(vwarp - vlive);
			if (!(dist <= distThres))
				return false;

			float sine = norm(cross(nwarp, nlive));
			if (!(sine < angleThres))
				return false;
#endif

			vl = Tbx::Point3(vlive.x, vlive.y, vlive.z);

			return true;
		}

		__device__ __forceinline__ void calc_dataterm () const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			vl = Tlw_inv * vl;

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[KnnK];

				// dqk_0
				{
					Tbx::Vec3 r, t;
					get_twist(knn_k(knn, 0), r, t);
					float4 nodeVw = get_nodesVw(knn_k(knn, 0));
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					dqk_0.from_twist(r, t);
					float expIn = nodesV.dot(nodesV) * nodeVw.w * nodeVw.w;
					wk[0] = __expf(-0.5f * expIn);
					dq = dq + dqk_0 * wk[0];
				}

				// other dqk_k
#pragma unroll
				for (int k = 1; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId >= nNodes)
						break;
					
					Tbx::Vec3 r, t;
					get_twist(knnNodeId, r, t);
					float4 nodeVw = get_nodesVw(knnNodeId);
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw))-v);
					Tbx::Dual_quat_cu dqk_k;
					dqk_k.from_twist(r, t);
#ifdef ENABLE_ANTI_PODALITY
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
						 *sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
#else
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w);
#endif
					dq = dq + dqk_k * wk[k];		
				}

				Tbx::Dual_quat_cu dq_bar = dq;
				float norm_dq_bar = dq_bar.norm();
				if (norm_dq_bar < Tbx::Dual_quat_cu::epsilon())
					return;
				float inv_norm_dq_bar = 1.f / norm_dq_bar;

				dq = dq * inv_norm_dq_bar; // normalize

				// the grad energy f
				const float f = data_term_penalty(dq.rotate(n).dot(dq.transform(v) - vl));

				// paitial_f_partial_T
				const Tbx::Transfo p_f_p_T = compute_p_f_p_T(n, v, vl, dq);

				for (int knnK = 0; knnK < KnnK; knnK++)
				{
					int knnNodeId = knn_k(knn, knnK);
					if (knnNodeId >= nNodes)
						break;
					float p_f_p_alpha[VarPerNode];
					float wk_k = wk[knnK] * inv_norm_dq_bar * 2;
					
					//// comput partial_T_partial_alphak, hard code here.
					Tbx::Dual_quat_cu p_qk_p_alpha;
					Tbx::Transfo p_T_p_alphak;
					Tbx::Vec3 t, r;
					float b, c;
					Tbx::Quat_cu q1;
					get_twist(knnNodeId, r, t);
					{
						float n = r.norm();
						float sin_n, cos_n;
						sincos(n, &sin_n, &cos_n);
						b = n > Tbx::Dual_quat_cu::epsilon() ? sin_n / n : 1;
						c = n > Tbx::Dual_quat_cu::epsilon() ? (cos_n - b) / (n*n) : 0;
						q1 = Tbx::Quat_cu(cos_n*0.5f, r.x*b*0.5f, r.y*b*0.5f, r.z*b*0.5f);
					}

					// alpha0
					p_qk_p_alpha[0] = -r[0] * b;
					p_qk_p_alpha[1] = b + r[0] * r[0] * c;
					p_qk_p_alpha[2] = r[0] * r[1] * c;
					p_qk_p_alpha[3] = r[0] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[0] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha1
					p_qk_p_alpha[0] = -r[1] * b;
					p_qk_p_alpha[1] = r[1] * r[0] * c;
					p_qk_p_alpha[2] = b + r[1] * r[1] * c;
					p_qk_p_alpha[3] = r[1] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[1] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha2
					p_qk_p_alpha[0] = -r[2] * b;
					p_qk_p_alpha[1] = r[2] * r[0] * c;
					p_qk_p_alpha[2] = r[2] * r[1] * c;
					p_qk_p_alpha[3] = b + r[2] * r[2] * c;
					p_qk_p_alpha = Tbx::Dual_quat_cu::dual_quat_from(p_qk_p_alpha.get_non_dual_part(), t);
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[2] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha3
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[1], q1[0], -q1[3], q1[2]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[3] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha4
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[2], q1[3], q1[0], -q1[1]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[4] = trace_AtB(p_f_p_T, p_T_p_alphak);

					// alpha5
					p_qk_p_alpha = Tbx::Dual_quat_cu(Tbx::Quat_cu(0, 0, 0, 0),
						Tbx::Quat_cu(-q1[3], -q1[2], q1[1], q1[0]));
					p_T_p_alphak = p_T_p_alphak_func(p_qk_p_alpha, dq_bar, dq,
						inv_norm_dq_bar, wk_k);
					p_f_p_alpha[5] = trace_AtB(p_f_p_T, p_T_p_alphak);

					//// reduce--------------------------------------------------
					int shift = knnNodeId * VarPerNode2;
					int shift_g = knnNodeId * VarPerNode;
					for (int i = 0; i < VarPerNode; ++i)
					{
#pragma unroll
						for (int j = 0; j <= i; ++j)
							atomicAdd(&Hd_[shift + j], p_f_p_alpha[i] * p_f_p_alpha[j]);
						atomicAdd(&g_[shift_g + i], p_f_p_alpha[i] * f);
						shift += VarPerNode;
					}// end for i					
				}// end for knnK
			}// end if found corr
		}// end function ()

		__device__ __forceinline__ Tbx::Dual_quat_cu calc_pixel_dq(KnnIdx knn, 
			Tbx::Point3 v, float* wk)const
		{
			Tbx::Dual_quat_cu dqk_0;
			Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0,0,0,0), Tbx::Quat_cu(0,0,0,0));
			// dqk_0
			{
				Tbx::Vec3 r, t;
				get_twist(knn_k(knn, 0), r, t);
				float4 nodeVw = get_nodesVw(knn_k(knn, 0));
				Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
				dqk_0.from_twist(r, t);
				wk[0] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w);
				dq += dqk_0 * wk[0];
			}

			// other dqk_k
#pragma unroll
			for (int k = 1; k < KnnK; k++)
			{
				if (knn_k(knn, k) >= nNodes)
					break;
				float4 nodeVw = get_nodesVw(knn_k(knn, k));
				Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
				Tbx::Dual_quat_cu dqk_k;
				Tbx::Vec3 r, t;
				get_twist(knn_k(knn, k), r, t);
				dqk_k.from_twist(r, t);
#ifdef ENABLE_ANTI_PODALITY
				wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
					*sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
#else
				wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w);
#endif
				dq += dqk_k * wk[k];
			}
			return dq;
		}

		__device__ __forceinline__ void exchange_ri_k(KnnIdx knn, 
			const float* wk, int k, int i, Tbx::Dual_quat_cu& dq, float& inc)const
		{
			Tbx::Vec3 r, t;
			get_twist(knn_k(knn, k), r, t);
			Tbx::Dual_quat_cu old_dqk, new_dqk;
			old_dqk.from_twist(r, t);
			inc = get_numeric_inc(r[i]);
			r[i] += inc;
			new_dqk.from_twist(r, t);
			dq -= old_dqk * wk[k];
			dq += new_dqk * wk[k] * sign(old_dqk.get_non_dual_part().dot(new_dqk.get_non_dual_part()));
		}
		__device__ __forceinline__ void exchange_ti_k(KnnIdx knn,
			const float* wk, int k, int i, Tbx::Dual_quat_cu& dq, float& inc)const
		{
			Tbx::Vec3 r, t;
			get_twist(knn_k(knn, k), r, t);
			Tbx::Dual_quat_cu old_dqk, new_dqk;
			old_dqk.from_twist(r, t);
			inc = get_numeric_inc(t[i]);
			t[i] += inc;
			new_dqk.from_twist(r, t);
			dq -= old_dqk * wk[k];
			dq += new_dqk * wk[k] * sign(old_dqk.get_non_dual_part().dot(new_dqk.get_non_dual_part()));
		}

		__device__ __forceinline__ float get_numeric_inc(float v) const
		{
			return max( 1e-5f, v* 1e-3f);
		}

		__device__ __forceinline__ void calc_dataterm_numeric() const
		{
			const int x = threadIdx.x + blockIdx.x * blockDim.x;
			const int y = threadIdx.y + blockIdx.y * blockDim.y;
			if (x >= imgWidth || y >= imgHeight)
				return;

			const KnnIdx knn = vmapKnn(y, x);
			Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
			Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

			if (isnan(n.x) || isnan(v.x))
				return;

			// 1. get all nodes params
			// 2. compute function=================================================
			float wk[KnnK];
			Tbx::Dual_quat_cu dq = calc_pixel_dq(knn, v, wk);
			float norm_dq = dq.norm();
			if (norm_dq < Tbx::Dual_quat_cu::epsilon())
				return;
			Tbx::Dual_quat_cu dq_not_normalized = dq;
			dq = dq * (1.f / norm_dq); // normalize

			// find corr
			Tbx::Vec3 nwarp = Tlw*dq.rotate(n);
			Tbx::Point3 vwarp = Tlw*dq.transform(v);
			Tbx::Point3 vl;
			//bool corr_found = search(convert(vwarp), convert(nwarp), vl);
			bool corr_found = search(x, y, vl);
			if (!corr_found)
				return;

			// the grad energy
			const float f = nwarp.dot(vwarp - vl);
			const float psi_f = data_term_penalty(f);

			// 3. compute jacobi
			for (int knnK = 0; knnK < KnnK; knnK++)
			{
				if (knn_k(knn, knnK) >= nNodes)
					break;
				float df[6];

				// 3.0 p_r[0:2]
				for (int i = 0; i < 3; i++)
				{
					float inc;
					Tbx::Dual_quat_cu dq1 = dq_not_normalized;
					exchange_ri_k(knn, wk, knnK, i, dq1, inc);
					dq1 *= (1.f / dq1.norm());
					nwarp = Tlw*dq1.rotate(n);
					vwarp = Tlw*dq1.transform(v);

					Tbx::Point3 vl1 = vl;
					//corr_found = search(convert(vwarp), convert(nwarp), vl1);
					//if (!corr_found)
					//	return;

					float f1 = nwarp.dot(vwarp - vl1);
					df[i] = (f1 - f) / inc;
				}// i=0:3

				// 3.1 p_t[0:2]
				for (int i = 0; i < 3; i++)
				{
					float inc;
					Tbx::Dual_quat_cu dq1 = dq_not_normalized;
					exchange_ti_k(knn, wk, knnK, i, dq1, inc);
					dq1 *= (1.f / dq1.norm());
					nwarp = Tlw*dq1.rotate(n);
					vwarp = Tlw*dq1.transform(v);

					Tbx::Point3 vl1 = vl;
					//corr_found = search(convert(vwarp), convert(nwarp), vl1);
					//if (!corr_found)
					//	return;

					float f1 = nwarp.dot(vwarp - vl1);
					df[i+3] = (f1 - f) / inc;
				}// i=0:3

				//// reduce--------------------------------------------------
				int shift = knn_k(knn, knnK) * VarPerNode2;
				int shift_g = knn_k(knn, knnK) * VarPerNode;
				for (int i = 0; i < VarPerNode; ++i)
				{
#pragma unroll
					for (int j = 0; j <= i; ++j)
						atomicAdd(&Hd_[shift + j], df[i] * df[j]);
					atomicAdd(&g_[shift_g + i], df[i] * psi_f);
					shift += VarPerNode;
				}// end for i
			}// end for knnK
		}// end function ()

		__device__ __forceinline__ void calcTotalEnergy()const
		{
			const int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
			const int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

			Tbx::Point3 vl;
			bool found_coresp = false;
			if (x < imgWidth && y < imgHeight)
				found_coresp = search(x, y, vl);

			if (found_coresp)
			{
				Tbx::Point3 v(convert(read_float3_4(vmap_cano(y, x))));
				Tbx::Vec3 n(convert(read_float3_4(nmap_cano(y, x))));

				const KnnIdx knn = vmapKnn(y, x);
				Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
				Tbx::Dual_quat_cu dqk_0;
				float wk[KnnK];
				// dqk_0
				{
					Tbx::Vec3 r, t;
					get_twist(knn_k(knn, 0), r, t);
					float4 nodeVw = get_nodesVw(knn_k(knn, 0));
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					dqk_0.from_twist(r, t);
					float expIn = nodesV.dot(nodesV) * nodeVw.w * nodeVw.w;
					wk[0] = __expf(-0.5f * expIn);
					dq = dq + dqk_0 * wk[0];
				}

				// other dqk_k
#pragma unroll
				for (int k = 1; k < KnnK; k++)
				{
					int knnNodeId = knn_k(knn, k);
					if (knnNodeId >= nNodes)
						break;

					Tbx::Vec3 r, t;
					get_twist(knnNodeId, r, t);
					float4 nodeVw = get_nodesVw(knnNodeId);
					Tbx::Vec3 nodesV(convert(read_float3_4(nodeVw)) - v);
					Tbx::Dual_quat_cu dqk_k;
					dqk_k.from_twist(r, t);
#ifdef ENABLE_ANTI_PODALITY
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w)
						*sign(dqk_0.get_non_dual_part().dot(dqk_k.get_non_dual_part()));
#else
					wk[k] = __expf(-0.5f * nodesV.dot(nodesV) * nodeVw.w * nodeVw.w);
#endif
					dq = dq + dqk_k * wk[k];
				}

				float norm_dq = dq.norm();
				if (norm_dq < Tbx::Dual_quat_cu::epsilon())
					return;
				dq = dq * (1.f / norm_dq); // normalize

				// the grad energy f
				const float f = data_term_energy((Tlw*dq.rotate(n)).dot(Tlw*dq.transform(v) - vl));
				//atomicAdd(totalEnergy, f);
				totalEnergy[y*imgWidth + x] = f;
			}//end if find corr
#ifdef DEBUG_ASSIGN_BIG_ENERGY_TO_NO_CORR
			else // debug: add constant penalty
			{
				totalEnergy[y*imgWidth + x] = data_term_energy(psi_data);
			}
#endif
		}
	};

	__global__ void dataTermCombinedKernel(const DataTermCombined cs)
	{
#ifdef CALC_DATA_TERM_NUMERIC
		cs.calc_dataterm_numeric();
#else
		cs.calc_dataterm();
#endif
	}

	void GpuGaussNewtonSolver::calcDataTerm()
	{
		DataTermCombined cs;
		cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
		cs.distThres = m_param->fusion_nonRigid_distThre;
		cs.Hd_ = m_Hd.value();
		cs.g_ = m_g;
		cs.imgHeight = m_vmap_cano->rows();
		cs.imgWidth = m_vmap_cano->cols();
		cs.intr = m_intr;
		cs.nmap_cano = *m_nmap_cano;
		cs.nmap_live = *m_nmap_live;
		cs.nmap_warp = *m_nmap_warp;
		cs.vmap_cano = *m_vmap_cano;
		cs.vmap_live = *m_vmap_live;
		cs.vmap_warp = *m_vmap_warp;
		cs.vmapKnn = m_vmapKnn;
		cs.nNodes = m_numNodes;
		cs.Tlw = m_pWarpField->get_rigidTransform();
		cs.Tlw_inv = m_pWarpField->get_rigidTransform().fast_invert();
		cs.psi_data = m_param->fusion_psi_data;

		//////////////////////////////
		dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
		dim3 grid(1, 1, 1);
		grid.x = divUp(cs.imgWidth, block.x);
		grid.y = divUp(cs.imgHeight, block.y);
		dataTermCombinedKernel<< <grid, block >> >(cs);
		cudaSafeCall(hipGetLastError(), "dataTermCombinedKernel");
	}

	__global__ void calcDataTermTotalEnergyKernel(const DataTermCombined cs)
	{
		cs.calcTotalEnergy();
	}

#pragma endregion

#pragma region --define sparse structure
	__global__ void count_Jr_rows_kernel(int* rctptr, int nMaxNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i >= nMaxNodes)
			return;
	
		KnnIdx knn = get_nodesKnn(i);
		int numK = -1;
		for (int k = 0; k < KnnK; ++k)
		{
			if (knn_k(knn, k) < nMaxNodes)
				numK = k;
		}

		// each node generate 6*maxK rows
		rctptr[i] = (numK + 1);
		
		if (i == 0)
			rctptr[nMaxNodes] = 0;
	}

	__global__ void compute_row_map_kernel(GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		const int* rctptr, int nMaxNodes)
	{
		int iNode = threadIdx.x + blockIdx.x*blockDim.x;
		if (iNode < nMaxNodes)
		{
			int row_b = rctptr[iNode];
			int row_e = rctptr[iNode+1];
			for (int r = row_b; r < row_e; r++)
			{
				GpuGaussNewtonSolver::JrRow2NodeMapper mp;
				mp.nodeId = iNode;
				mp.k = r - row_b;
				mp.ixyz = 0;
				row2nodeId[r] = mp;
			}
		}
	}

	__global__ void compute_Jr_rowPtr_kernel(
		int* rptr, const GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId,
		int nMaxNodes, int nBlockRows)
	{
		enum{
			BlocksPerRow = 2
		};
		const int iBlockRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iBlockRow >= nBlockRows)
			return;

		const int iNode = row2nodeId[iBlockRow].nodeId;
		if (iNode < nMaxNodes)
		{
			KnnIdx knn = get_nodesKnn(iNode);
			if (knn_k(knn, row2nodeId[iBlockRow].k) < nMaxNodes)
				rptr[iBlockRow] = iBlockRow * BlocksPerRow;
		}

		// the 1st thread also write the last value
		if (iBlockRow == 0)
			rptr[nBlockRows] = nBlockRows * BlocksPerRow;
	}

	__global__ void compute_Jr_colIdx_kernel(
		int* colIdx, const GpuGaussNewtonSolver::JrRow2NodeMapper* row2nodeId, 
		int nMaxNodes, int nBlockRows)
	{
		enum{
			ColPerRow = 2
		};
		const int iBlockRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iBlockRow >= nBlockRows)
			return;

		const int iNode = row2nodeId[iBlockRow].nodeId;
		if (iNode < nMaxNodes)
		{
			KnnIdx knn = get_nodesKnn(iNode);
			int knnNodeId = knn_k(knn, row2nodeId[iBlockRow].k);
			if (knnNodeId < nMaxNodes)
			{
				int col_b = iBlockRow*ColPerRow;

				// each row 2 blocks
				// 1. self
				colIdx[col_b] = iNode;

				// 2. neighbor
				colIdx[col_b + 1] = knnNodeId;
			}// end if knnNodeId
		}
	}

	__global__ void calc_B_cidx_kernel(int* B_cidx, 
		const int* B_rptr, int nBlockInRows, int nMaxNodes, int nLv0Nodes)
	{
		int iBlockRow = threadIdx.x + blockIdx.x*blockDim.x;
		if (iBlockRow < nBlockInRows)
		{
			KnnIdx knn = get_nodesKnn(iBlockRow);
			int col_b = B_rptr[iBlockRow];
			for (int k = 0; k < KnnK; ++k)
			{
				int knnNodeId = knn_k(knn, k);
				if (knnNodeId < nMaxNodes)
					B_cidx[col_b++] = knnNodeId-nLv0Nodes;
			}
		}
	}

	void GpuGaussNewtonSolver::initSparseStructure()
	{
		// 1. compute Jr structure ==============================================
		// 1.0. decide the total rows we have for each nodes
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			count_Jr_rows_kernel << <grid, block >> >(m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::count_Jr_rows_kernel");
			thrust_wrapper::exclusive_scan(m_Jr_RowCounter.ptr(), m_Jr_RowCounter.ptr(), m_numNodes + 1);
			int jrRows = 0;
			cudaSafeCall(hipMemcpy(&jrRows, m_Jr_RowCounter.ptr() + m_numNodes,
				sizeof(int), hipMemcpyDeviceToHost), "copy Jr rows to host");
			m_Jr->resize(jrRows, m_numNodes, RowPerNode_RegTerm, VarPerNode);
		}

		// 1.1. collect nodes edges info:
		//	each low-level nodes are connected to k higher level nodes
		//	but the connections are not stored for the higher level nodes
		//  thus when processing each node, we add 2*k edges, w.r.t. 2*k*3 rows: each (x,y,z) a row
		//	for each row, there are exactly 2*VarPerNode values
		//	after this step, we can get the CSR/COO structure
		if (m_Jr->rows() > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_numNodes, block.x));
			compute_row_map_kernel << <grid, block >> >(m_Jr_RowMap2NodeId.ptr(), m_Jr_RowCounter.ptr(), m_numNodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_row_map_kernel");
		}
		if (m_Jr->rows() > 0)
		{
			m_Jr->beginConstructRowPtr();
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jr->blocksInRow(), block.x));
			compute_Jr_rowPtr_kernel << <grid, block >> >(m_Jr->bsrRowPtr(),
				 m_Jr_RowMap2NodeId.ptr(), m_numNodes, m_Jr->blocksInRow());
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_Jr_rowPtr_kernel");
			m_Jr->endConstructRowPtr();

			compute_Jr_colIdx_kernel << <grid, block >> >(m_Jr->bsrColIdx(), 
				m_Jr_RowMap2NodeId.ptr(), m_numNodes, m_Jr->blocksInRow());
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::compute_Jr_colIdx_kernel");
		}

		// 2. compute Jrt structure ==============================================
		// 2.1. fill (row, col) as (col, row) from Jr and sort.
		m_Jr->transposeStructureTo(*m_Jrt);
		m_Jrt->subRows_structure(*m_Jrt13_structure, m_numLv0Nodes, m_numNodes);
		m_Jrt13_structure->transposeStructureTo(*m_Jr13_structure);
		m_Jrt13_structure->multBsr_structure(*m_Jr13_structure, *m_Hr);

		// 3. compute B structure ==============================================
		// 3.1 the row ptr of B is the same CSR info with the first L0 rows of Jrt.
		m_B->resize(m_numLv0Nodes, m_Jr->blocksInCol() - m_numLv0Nodes, VarPerNode, VarPerNode);
		m_B->setRowFromBsrRowPtr(m_Jrt->bsrRowPtr());
		
		// 3.2 the col-idx of B
		if (m_B->rows() > 0)
		{
			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_B->blocksInRow(), block.x));
			calc_B_cidx_kernel << <grid, block >> >(
				m_B->bsrColIdx(), m_B->bsrRowPtr(), m_B->blocksInRow(), m_numNodes, m_numLv0Nodes);
			cudaSafeCall(hipGetLastError(), "GpuGaussNewtonSolver::initSparseStructure::calc_B_cidx_kernel");
		}

		// 3.3 sort to compute Bt
		m_B->transposeStructureTo(*m_Bt);

		m_Hd.resize(m_numLv0Nodes, VarPerNode);
		m_Hd_Linv.resize(m_numLv0Nodes, VarPerNode);
		m_Hd_LLtinv.resize(m_numLv0Nodes, VarPerNode);
		m_Bt->rightMultDiag_structure(m_Hd_Linv, *m_Bt_Ltinv);

		// 4. single level Hessian
		if (m_param->graph_single_level)
		{
			m_Jrt->multBsr_structure(*m_Jr, *m_H_singleLevel);
			m_singleLevel_solver->analysis(m_H_singleLevel, true);
		}
		else
		{
			// sovle Q on CPU, prepare for it
			m_Bt->multBsr_structure(*m_B, *m_Q, m_Hr);
			m_singleLevel_solver->analysis(m_Q, true);
		}
	}

#pragma endregion

#pragma region --calc reg term
	struct RegTermJacobi
	{
		typedef GpuGaussNewtonSolver::JrRow2NodeMapper Mapper;
		enum
		{
			VarPerNode = GpuGaussNewtonSolver::VarPerNode,
			VarPerNode2 = VarPerNode*VarPerNode,
			ColPerRow = VarPerNode * 2
		};

		int nNodes;
		int nBlockRows;
		const Mapper* rows2nodeIds;
		const int* rptr;
		mutable float* vptr;
		mutable float* fptr;

		int nNodesEachLevel[WarpField::GraphLevelNum];
		float dw_scale_each_level;
		float dw_softness;

		float psi_reg;
		float lambda;

		float* totalEnergy;


		__device__ __forceinline__ int getNodeLevel(int nodeId)const
		{
			for (int k = 0; k < WarpField::GraphLevelNum; k++)
			if (nodeId < nNodesEachLevel[k])
				return k;
			return WarpField::GraphLevelNum;
		}

		__device__ __forceinline__ float calc_alpha_reg(int nodeId, int k, int nMaxNodes)const
		{
			KnnIdx knn = get_nodesKnn(nodeId);

			float4 nodeVwi = get_nodesVw(nodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			float4 nodeVwj = get_nodesVw(knn_k(knn, k));
			float invW = min(nodeVwi.w, nodeVwj.w);

			float wk = 0.f, sum_w = 0.f;
			for (int knn_idx = 0; knn_idx < KnnK; knn_idx++)
			{
				if (knn_idx < nMaxNodes)
				{
					float4 nodeVwj = get_nodesVw(knn_k(knn, knn_idx));
					Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
					float w = __expf(-dw_softness * (vi - vj).dot(vi - vj) * invW * invW);
					sum_w += w;
					if (knn_idx == k)
						wk = w;
				}
			}

			// if all neighbors are too far to give valid weightings, 
			// we just take an average.
			if (sum_w < 1e-6f)
				wk = 0.25f;
			else
				wk /= sum_w;

			return wk * __powf(dw_scale_each_level, getNodeLevel(nodeId));
		}

		__device__ __forceinline__  Tbx::Dual_quat_cu p_qk_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Vec3 t, r;
			float b, c, n;
			Tbx::Quat_cu q0(0, 0, 0, 0), q1 = dq.get_non_dual_part();
			switch (i)
			{
			case 0:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.x * b;
					q0.coeff1 = b + r.x*r.x*c;
					q0.coeff2 = r.x*r.y*c;
					q0.coeff3 = r.x*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 1;
					q0.coeff2 = 0;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 1:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);
					q0.coeff0 = -r.y * b;
					q0.coeff1 = r.y*r.x*c;
					q0.coeff2 = b + r.y*r.y*c;
					q0.coeff3 = r.y*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 1;
					q0.coeff3 = 0;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 2:
				dq.to_twist(r, t);
				n = r.norm();
				if (n > Tbx::Dual_quat_cu::epsilon())
				{
					b = sin(n) / n;
					c = (cos(n) - b) / (n*n);

					q0.coeff0 = -r.z * b;
					q0.coeff1 = r.z*r.x*c;
					q0.coeff2 = r.z*r.y*c;
					q0.coeff3 = b + r.z*r.z*c;
				}
				else
				{
					q0.coeff0 = 0;
					q0.coeff1 = 0;
					q0.coeff2 = 0;
					q0.coeff3 = 1;
				}

				q1.coeff0 = (t.x * q0.coeff1 + t.y * q0.coeff2 + t.z * q0.coeff3) * (-0.5);
				q1.coeff1 = (t.x * q0.coeff0 + t.y * q0.coeff3 - t.z * q0.coeff2) * 0.5;
				q1.coeff2 = (-t.x * q0.coeff3 + t.y * q0.coeff0 + t.z * q0.coeff1) * 0.5;
				q1.coeff3 = (t.x * q0.coeff2 - t.y * q0.coeff1 + t.z * q0.coeff0) * 0.5;
				return Tbx::Dual_quat_cu(q0, q1);
			case 3:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff1, q1.coeff0, -q1.coeff3, q1.coeff2))*0.5;
			case 4:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff2, q1.coeff3, q1.coeff0, -q1.coeff1))*0.5;
			case 5:
				return Tbx::Dual_quat_cu(q0, Tbx::Quat_cu(-q1.coeff3, -q1.coeff2, q1.coeff1, q1.coeff0))*0.5;
			default:
				return Tbx::Dual_quat_cu();
			}
		}

		__device__ __forceinline__  float reg_term_energy(Tbx::Vec3 f)const
		{
#ifdef USE_L2_NORM_REG_TERM
			return 0.5f*f.dot(f);
#else
			// the robust Huber penelty gradient
			float s = 0;
			float norm = f.norm();
			if (norm < psi_reg)
				s = norm * norm * 0.5f;
			else
				s = psi_reg*(norm - psi_reg*0.5f);
			return s;
#endif
		}

		__device__ __forceinline__  Tbx::Vec3 reg_term_penalty(Tbx::Vec3 f)const
		{
#ifdef USE_L2_NORM_REG_TERM
			return f;
#else
			// the robust Huber penelty gradient
			Tbx::Vec3 df;
			float norm = f.norm();
			if (norm < psi_reg)
				df = f;
			else
			for (int k = 0; k < 3; k++)
				df[k] = f[k]*psi_reg / norm;
			return df;
#endif
		}

		__device__ __forceinline__  Tbx::Transfo p_SE3_p_alpha_func(Tbx::Dual_quat_cu dq, int i)const
		{
			Tbx::Transfo T = Tbx::Transfo::empty();
			Tbx::Dual_quat_cu p_dq_p_alphai = p_qk_p_alpha_func(dq, i) * 2.f;

			//// evaluate p_dqi_p_alphak, heavily hard code here
			//// this hard code is crucial to the performance 
			// 0:
			// (0, -z0, y0, x1,
			// z0, 0, -x0, y1,
			//-y0, x0, 0, z1,
			// 0, 0, 0, 0) * 2;
			float p_dqi_p_alphak = p_dq_p_alphai[0];
			T[1] += -dq[3] * p_dqi_p_alphak;
			T[2] += dq[2] * p_dqi_p_alphak;
			T[3] += dq[5] * p_dqi_p_alphak;
			T[4] += dq[3] * p_dqi_p_alphak;
			T[6] += -dq[1] * p_dqi_p_alphak;
			T[7] += dq[6] * p_dqi_p_alphak;
			T[8] += -dq[2] * p_dqi_p_alphak;
			T[9] += dq[1] * p_dqi_p_alphak;
			T[11] += dq[7] * p_dqi_p_alphak;

			// 1
			//( 0, y0, z0, -w1,
			//	y0, -2 * x0, -w0, -z1,
			//	z0, w0, -2 * x0, y1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[1];
			T[1] += dq[2] * p_dqi_p_alphak;
			T[2] += dq[3] * p_dqi_p_alphak;
			T[3] += -dq[4] * p_dqi_p_alphak;
			T[4] += dq[2] * p_dqi_p_alphak;
			T[5] += -dq[1] * p_dqi_p_alphak * 2;
			T[6] += -dq[0] * p_dqi_p_alphak;
			T[7] += -dq[7] * p_dqi_p_alphak;
			T[8] += dq[3] * p_dqi_p_alphak;
			T[9] += dq[0] * p_dqi_p_alphak;
			T[10] += -dq[1] * p_dqi_p_alphak * 2;
			T[11] += dq[6] * p_dqi_p_alphak;

			// 2.
			// (-2 * y0, x0, w0, z1,
			//	x0, 0, z0, -w1,
			//	-w0, z0, -2 * y0, -x1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[2];
			T[0] += -dq[2] * p_dqi_p_alphak * 2;
			T[1] += dq[1] * p_dqi_p_alphak;
			T[2] += dq[0] * p_dqi_p_alphak;
			T[3] += dq[7] * p_dqi_p_alphak;
			T[4] += dq[1] * p_dqi_p_alphak;
			T[6] += dq[3] * p_dqi_p_alphak;
			T[7] += -dq[4] * p_dqi_p_alphak;
			T[8] += -dq[0] * p_dqi_p_alphak;
			T[9] += dq[3] * p_dqi_p_alphak;
			T[10] += -dq[2] * p_dqi_p_alphak * 2;
			T[11] += -dq[5] * p_dqi_p_alphak;

			// 3.
			// (-2 * z0, -w0, x0, -y1,
			//	w0, -2 * z0, y0, x1,
			//	x0, y0, 0, -w1,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[3];
			T[0] += -dq[3] * p_dqi_p_alphak * 2;
			T[1] += -dq[0] * p_dqi_p_alphak;
			T[2] += dq[1] * p_dqi_p_alphak;
			T[3] += -dq[6] * p_dqi_p_alphak;
			T[4] += dq[0] * p_dqi_p_alphak;
			T[5] += -dq[3] * p_dqi_p_alphak * 2;
			T[6] += dq[2] * p_dqi_p_alphak;
			T[7] += dq[5] * p_dqi_p_alphak;
			T[8] += dq[1] * p_dqi_p_alphak;
			T[9] += dq[2] * p_dqi_p_alphak;
			T[11] += -dq[4] * p_dqi_p_alphak;

			// 4.
			//( 0, 0, 0, -x0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, -z0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[4];
			T[3] += -dq[1] * p_dqi_p_alphak;
			T[7] += -dq[2] * p_dqi_p_alphak;
			T[11] += -dq[3] * p_dqi_p_alphak;

			// 5. 
			// (0, 0, 0, w0,
			//	0, 0, 0, z0,
			//	0, 0, 0, -y0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[5];
			T[3] += dq[0] * p_dqi_p_alphak;
			T[7] += dq[3] * p_dqi_p_alphak;
			T[11] += -dq[2] * p_dqi_p_alphak;

			// 6. 
			// (0, 0, 0, -z0,
			//	0, 0, 0, w0,
			//	0, 0, 0, x0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[6];
			T[3] += -dq[3] * p_dqi_p_alphak;
			T[7] += dq[0] * p_dqi_p_alphak;
			T[11] += dq[1] * p_dqi_p_alphak;

			// 7.
			// (0, 0, 0, y0,
			//	0, 0, 0, -x0,
			//	0, 0, 0, w0,
			//	0, 0, 0, 0) * 2;
			p_dqi_p_alphak = p_dq_p_alphai[7];
			T[3] += dq[2] * p_dqi_p_alphak;
			T[7] += -dq[1] * p_dqi_p_alphak;
			T[11] += dq[0] * p_dqi_p_alphak;

			return T;
		}

		__device__ __forceinline__ void operator () () const
		{
			const int iBlockRow = threadIdx.x + blockIdx.x * blockDim.x;
		
			if (iBlockRow >= nBlockRows)
				return;

			Mapper mapper = rows2nodeIds[iBlockRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = calc_alpha_reg(mapper.nodeId, mapper.k, nNodes);
			float ww = sqrt(lambda * alpha_ij);

			//if (isinf(nodeVwj.w))
			//	printf("inf found: %d %d %f %f %f %f\n", mapper.nodeId, knnNodeId, 
			//	nodeVwj.w, 1.f / nodeVwj.w, alpha_ij, ww);
			
			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			val = reg_term_penalty(val);
			
			const int iRow = iBlockRow * RowPerNode_RegTerm;
			fptr[iRow + 0] = val.x * ww;
			fptr[iRow + 1] = val.y * ww;
			fptr[iRow + 2] = val.z * ww;

#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val1 = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
			val1 = reg_term_penalty(val1);
			fptr[iRow + 3] = val1.x * ww;
			fptr[iRow + 4] = val1.y * ww;
			fptr[iRow + 5] = val1.z * ww;
#endif

			// jacobi=============================================
			int cooPos0 = rptr[iBlockRow] * RowPerNode_RegTerm * VarPerNode;
			int cooPos1 = cooPos0 + RowPerNode_RegTerm * VarPerNode;
			for (int ialpha = 0; ialpha < VarPerNode; ialpha++)
			{
				Tbx::Transfo p_Ti_p_alpha = p_SE3_p_alpha_func(dqi, ialpha);
				Tbx::Transfo p_Tj_p_alpha = p_SE3_p_alpha_func(dqj, ialpha);

				// partial_psi_partial_alpha
				Tbx::Vec3 p_psi_p_alphai_j = (p_Ti_p_alpha * vj) * ww;
				Tbx::Vec3 p_psi_p_alphaj_j = (p_Tj_p_alpha * vj) * (-ww);
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
				Tbx::Vec3 p_psi_p_alphai_i = (p_Ti_p_alpha * vi) * (-ww);
				Tbx::Vec3 p_psi_p_alphaj_i = (p_Tj_p_alpha * vi) * ww;
#endif

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					vptr[cooPos0 + ixyz*VarPerNode + ialpha] = p_psi_p_alphai_j[ixyz];
					vptr[cooPos1 + ixyz*VarPerNode + ialpha] = p_psi_p_alphaj_j[ixyz];
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					vptr[cooPos0 + (3 + ixyz)*VarPerNode + ialpha] = p_psi_p_alphai_i[ixyz];
					vptr[cooPos1 + (3 + ixyz)*VarPerNode + ialpha] = p_psi_p_alphaj_i[ixyz];
#endif
				}
			}// end for ialpha
		}// end function ()

		__device__ __forceinline__ float get_numeric_inc(float v) const
		{
			return max(1e-5f, v* 1e-3f);
		}

		__device__ __forceinline__ void calc_reg_numeric () const
		{
			const int iBlockRow = threadIdx.x + blockIdx.x * blockDim.x;

			if (iBlockRow >= nBlockRows)
				return;

			Mapper mapper = rows2nodeIds[iBlockRow];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = calc_alpha_reg(mapper.nodeId, mapper.k, nNodes);
			float ww = sqrt(lambda * alpha_ij);

			// energy=============================================
			Tbx::Vec3 val_j = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			Tbx::Vec3 psi_val_j = reg_term_penalty(val_j);

			const int iRow = iBlockRow * RowPerNode_RegTerm;
			fptr[iRow + 0] = psi_val_j.x * ww;
			fptr[iRow + 1] = psi_val_j.y * ww;
			fptr[iRow + 2] = psi_val_j.z * ww;

#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val_i = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
			Tbx::Vec3 psi_val_i = reg_term_penalty(val_i);
			fptr[iRow + 3] = psi_val_i.x * ww;
			fptr[iRow + 4] = psi_val_i.y * ww;
			fptr[iRow + 5] = psi_val_i.z * ww;
#endif

			// jacobi=============================================
			int cooPos0 = rptr[iBlockRow] * RowPerNode_RegTerm * VarPerNode;
			int cooPos1 = cooPos0 + RowPerNode_RegTerm * VarPerNode;
			for (int ialpha = 0; ialpha < 3; ialpha++)
			{
				float inci = get_numeric_inc(ri[ialpha]);
				ri[ialpha] += inci;
				dqi.from_twist(ri, ti);
				Tbx::Vec3 val_j_inci = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_inci = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				ri[ialpha] -= inci;
				dqi.from_twist(ri, ti);

				float incj = get_numeric_inc(rj[ialpha]);
				rj[ialpha] += incj;
				dqj.from_twist(rj, tj);
				Tbx::Vec3 val_j_incj = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_incj = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				rj[ialpha] -= incj;
				dqj.from_twist(rj, tj);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					vptr[cooPos0 + ixyz*VarPerNode + ialpha] = ww * (val_j_inci[ixyz] - val_j[ixyz]) / inci;
					vptr[cooPos1 + ixyz*VarPerNode + ialpha] = ww * (val_j_incj[ixyz] - val_j[ixyz]) / incj;
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					vptr[cooPos0 + (3 + ixyz)*VarPerNode + ialpha] = ww * (val_i_inci[ixyz] - val_i[ixyz]) / inci;
					vptr[cooPos1 + (3 + ixyz)*VarPerNode + ialpha] = ww * (val_i_incj[ixyz] - val_i[ixyz]) / incj;
#endif
				}
			}// end for ialpha
			cooPos0 += 3;
			cooPos1 += 3;
			for (int ialpha = 0; ialpha < 3; ialpha++)
			{
				float inci = get_numeric_inc(ti[ialpha]);
				ti[ialpha] += inci;
				dqi.from_twist(ri, ti);
				Tbx::Vec3 val_j_inci = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_inci = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				ti[ialpha] -= inci;
				dqi.from_twist(ri, ti);

				float incj = get_numeric_inc(tj[ialpha]);
				tj[ialpha] += incj;
				dqj.from_twist(rj, tj);
				Tbx::Vec3 val_j_incj = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
				Tbx::Vec3 val_i_incj = dqj.transform(Tbx::Point3(vi)) - dqi.transform(Tbx::Point3(vi));
				tj[ialpha] -= incj;
				dqj.from_twist(rj, tj);

				for (int ixyz = 0; ixyz < 3; ixyz++)
				{
					vptr[cooPos0 + ixyz*VarPerNode + ialpha] = ww * (val_j_inci[ixyz] - val_j[ixyz]) / inci;
					vptr[cooPos1 + ixyz*VarPerNode + ialpha] = ww * (val_j_incj[ixyz] - val_j[ixyz]) / incj;
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
					vptr[cooPos0 + (3 + ixyz)*VarPerNode + ialpha] = ww * (val_i_inci[ixyz] - val_i[ixyz]) / inci;
					vptr[cooPos1 + (3 + ixyz)*VarPerNode + ialpha] = ww * (val_i_incj[ixyz] - val_i[ixyz]) / incj;
#endif
				}
			}// end for ialpha
		}// end function ()

		__device__ __forceinline__ void calcTotalEnergy () const
		{
			const int iNode = threadIdx.x + blockIdx.x * blockDim.x;

			if (iNode >= nBlockRows)
				return;

			Mapper mapper = rows2nodeIds[iNode];
			int knnNodeId = knn_k(get_nodesKnn(mapper.nodeId), mapper.k);

			if (knnNodeId >= nNodes)
				return;

			Tbx::Dual_quat_cu dqi, dqj;
			Tbx::Vec3 ri, ti, rj, tj;
			get_twist(mapper.nodeId, ri, ti);
			get_twist(knnNodeId, rj, tj);
			dqi.from_twist(ri, ti);
			dqj.from_twist(rj, tj);

			float4 nodeVwi = get_nodesVw(mapper.nodeId);
			float4 nodeVwj = get_nodesVw(knnNodeId);
			Tbx::Point3 vi(convert(read_float3_4(nodeVwi)));
			Tbx::Point3 vj(convert(read_float3_4(nodeVwj)));
			float alpha_ij = max(1.f / nodeVwi.w, 1.f / nodeVwj.w);
			float ww2 = lambda * calc_alpha_reg(mapper.nodeId, mapper.k, nNodes);

			// energy=============================================
			Tbx::Vec3 val = dqi.transform(Tbx::Point3(vj)) - dqj.transform(Tbx::Point3(vj));
			float eg = ww2 * reg_term_energy(val);
#ifndef DEFINE_USE_HALF_GRAPH_EDGE
			Tbx::Vec3 val1 = dqi.transform(Tbx::Point3(vi)) - dqj.transform(Tbx::Point3(vi));
			eg += ww2 * reg_term_energy(val1);
#endif

			//atomicAdd(totalEnergy, eg);
			totalEnergy[iNode] = eg;
		}
	};

	__global__ void calcRegTerm_kernel(RegTermJacobi rj)
	{
#ifdef CALC_REG_TERM_NUMERIC
		rj.calc_reg_numeric();
#else
		rj();
#endif
	}
	__global__ void calcRegTermTotalEnergy_kernel(RegTermJacobi rj)
	{
		rj.calcTotalEnergy();
	}

	void GpuGaussNewtonSolver::calcRegTerm()
	{
		if (m_Jr->rows() > 0)
		{
			CHECK_LE(m_Jr->rows(), m_f_r.size());

			RegTermJacobi rj;
			rj.lambda = m_param->fusion_lambda;
			rj.nNodes = m_numNodes;
			rj.nBlockRows = m_Jr->blocksInRow();
			rj.psi_reg = m_param->fusion_psi_reg;
			rj.rows2nodeIds = m_Jr_RowMap2NodeId;
			rj.rptr = m_Jr->bsrRowPtr();
			rj.vptr = m_Jr->value();
			rj.fptr = m_f_r.ptr();
			for (int k = 0; k < WarpField::GraphLevelNum; k++)
				rj.nNodesEachLevel[k] = m_pWarpField->getNumNodesInLevel(k);
			for (int k = 1; k < WarpField::GraphLevelNum; k++)
				rj.nNodesEachLevel[k] += rj.nNodesEachLevel[k-1];
			rj.dw_scale_each_level = m_param->warp_param_dw_lvup_scale;
			rj.dw_softness = m_param->warp_param_softness;

			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jr->rows() / RowPerNode_RegTerm, block.x));

			calcRegTerm_kernel << <grid, block >> >(rj);
			cudaSafeCall(hipGetLastError(), "calcRegTerm_kernel");

			// 2. compute Jrt ==============================================
			// 2.1. fill (row, col) as (col, row) from Jr and sort.
			m_Jr->transposeValueTo(*m_Jrt);
		}
	}
#pragma endregion

#pragma region --calcTotalEnergy
	float GpuGaussNewtonSolver::calcTotalEnergy(float& data_energy, float& reg_energy)
	{
		float total_energy = 0.f;
		hipMemset(m_energy_vec.ptr(), 0, m_energy_vec.sizeBytes());
		{
			DataTermCombined cs;
			cs.angleThres = m_param->fusion_nonRigid_angleThreSin;
			cs.distThres = m_param->fusion_nonRigid_distThre;
			cs.Hd_ = m_Hd.value();
			cs.g_ = m_g;
			cs.imgHeight = m_vmap_cano->rows();
			cs.imgWidth = m_vmap_cano->cols();
			cs.intr = m_intr;
			cs.nmap_cano = *m_nmap_cano;
			cs.nmap_live = *m_nmap_live;
			cs.nmap_warp = *m_nmap_warp;
			cs.vmap_cano = *m_vmap_cano;
			cs.vmap_live = *m_vmap_live;
			cs.vmap_warp = *m_vmap_warp;
			cs.vmapKnn = m_vmapKnn;
			cs.nNodes = m_numNodes;
			cs.Tlw = m_pWarpField->get_rigidTransform();
			cs.Tlw_inv = m_pWarpField->get_rigidTransform().fast_invert();
			cs.psi_data = m_param->fusion_psi_data;
			cs.totalEnergy = m_energy_vec.ptr();

			//int zero_mem_symbol = 0;
			//hipMemcpyToSymbol(HIP_SYMBOL(g_totalEnergy), &zero_mem_symbol, sizeof(int));
			//hipMemset(&m_tmpvec[0], 0, sizeof(float));

			// 1. data term
			//////////////////////////////
			dim3 block(CTA_SIZE_X, CTA_SIZE_Y);
			dim3 grid(1, 1, 1);
			grid.x = divUp(cs.imgWidth, block.x);
			grid.y = divUp(cs.imgHeight, block.y);
			calcDataTermTotalEnergyKernel << <grid, block >> >(cs);
			cudaSafeCall(hipGetLastError(), "calcDataTermTotalEnergyKernel");
		}

		if (m_Jr->rows() > 0)
		{
			RegTermJacobi rj;
			rj.lambda = m_param->fusion_lambda;
			rj.nNodes = m_numNodes;
			rj.nBlockRows = m_Jr->blocksInRow();
			rj.psi_reg = m_param->fusion_psi_reg;
			rj.rows2nodeIds = m_Jr_RowMap2NodeId;
			rj.rptr = m_Jr->bsrRowPtr();
			rj.vptr = m_Jr->value();
			rj.fptr = m_f_r.ptr();
			rj.totalEnergy = m_energy_vec.ptr() + m_vmapKnn.rows()*m_vmapKnn.cols();
			for (int k = 0; k < WarpField::GraphLevelNum; k++)
				rj.nNodesEachLevel[k] = m_pWarpField->getNumNodesInLevel(k);
			for (int k = 1; k < WarpField::GraphLevelNum; k++)
				rj.nNodesEachLevel[k] += rj.nNodesEachLevel[k - 1];
			rj.dw_scale_each_level = m_param->warp_param_dw_lvup_scale;
			rj.dw_softness = m_param->warp_param_softness;

			dim3 block(CTA_SIZE);
			dim3 grid(divUp(m_Jr->rows() / RowPerNode_RegTerm, block.x));

			calcRegTermTotalEnergy_kernel << <grid, block >> >(rj);
			cudaSafeCall(hipGetLastError(), "calcRegTermTotalEnergy_kernel");
		}

		//cudaSafeCall(hipMemcpy(&total_energy,
		//	&m_tmpvec[0], sizeof(float), hipMemcpyDeviceToHost), "copy reg totalEnergy to host");
		hipblasStatus_t st = hipblasSasum(m_cublasHandle, m_Jr->rows() / RowPerNode_RegTerm + 
			m_vmapKnn.rows()*m_vmapKnn.cols(),
			m_energy_vec.ptr(), 1, &total_energy);
		if (st != HIPBLAS_STATUS_SUCCESS)
			throw std::exception("cublass error, in hipblasSnrm2");

		// debug get both data and reg term energy
#if 1
		reg_energy = 0.f;
		if (m_Jr->rows() > 0)
		{
			hipblasSasum(m_cublasHandle, m_Jr->rows() / RowPerNode_RegTerm,
				m_energy_vec.ptr() + m_vmapKnn.rows()*m_vmapKnn.cols(),
				1, &reg_energy);
		}
		data_energy = total_energy - reg_energy;
#endif

		return total_energy;
	}
#pragma endregion

#pragma region --update twist

	__global__ void updateTwist_inch_kernel(float* twist, const float* h, float step, int nNodes)
	{
		int i = threadIdx.x + blockIdx.x*blockDim.x;
		if (i < nNodes)
		{
			int i6 = i * 6;
			Tbx::Vec3 r(twist[i6] + step*h[i6], twist[i6 + 1] + step*h[i6 + 1], twist[i6 + 2] + step*h[i6 + 2]);
			Tbx::Vec3 t(twist[i6+3] + step*h[i6+3], twist[i6 + 4] + step*h[i6 + 4], twist[i6 + 5] + step*h[i6 + 5]);
			Tbx::Dual_quat_cu dq;
			dq.from_twist(r, t);
			dq.to_twist(r, t);
			twist[i6] = r[0];
			twist[i6 + 1] = r[1];
			twist[i6 + 2] = r[2];
			twist[i6 + 3] = t[0];
			twist[i6 + 4] = t[1];
			twist[i6 + 5] = t[2];
		}
	}

	void GpuGaussNewtonSolver::updateTwist_inch(const float* h, float step)
	{
		dim3 block(CTA_SIZE);
		dim3 grid(divUp(m_numNodes, block.x));
		updateTwist_inch_kernel << <grid, block >> >(m_twist.ptr(), h, step, m_numNodes);
		cudaSafeCall(hipGetLastError(), "updateTwist_inch_kernel");
	}
#pragma endregion

#pragma region --factor out rigid

	__device__ float _g_common_q[8];

	template<int CTA_SIZE_, typename T>
	static __device__ __forceinline__ void reduce(volatile T* buffer)
	{
		int tid = Block::flattenedThreadId();
		T val = buffer[tid];

		if (CTA_SIZE_ >= 1024) { if (tid < 512) buffer[tid] = val = val + buffer[tid + 512]; __syncthreads(); }
		if (CTA_SIZE_ >= 512) { if (tid < 256) buffer[tid] = val = val + buffer[tid + 256]; __syncthreads(); }
		if (CTA_SIZE_ >= 256) { if (tid < 128) buffer[tid] = val = val + buffer[tid + 128]; __syncthreads(); }
		if (CTA_SIZE_ >= 128) { if (tid <  64) buffer[tid] = val = val + buffer[tid + 64]; __syncthreads(); }

		if (tid < 32){
			if (CTA_SIZE_ >= 64) { buffer[tid] = val = val + buffer[tid + 32]; }
			if (CTA_SIZE_ >= 32) { buffer[tid] = val = val + buffer[tid + 16]; }
			if (CTA_SIZE_ >= 16) { buffer[tid] = val = val + buffer[tid + 8]; }
			if (CTA_SIZE_ >= 8) { buffer[tid] = val = val + buffer[tid + 4]; }
			if (CTA_SIZE_ >= 4) { buffer[tid] = val = val + buffer[tid + 2]; }
			if (CTA_SIZE_ >= 2) { buffer[tid] = val = val + buffer[tid + 1]; }
		}
	}

	__global__ void reduce_all_nodes_kernel(const float4* nodesDqVw, int n)
	{
		const float* beg = (const float*)nodesDqVw + blockIdx.x;
		float sum = 0.f;
		for (int i = threadIdx.x; i < n; i += blockDim.x)
			sum += beg[i * 12]; // dq+vw, 12 float per node

		__shared__ float smem[GpuGaussNewtonSolver::CTA_SIZE];

		smem[threadIdx.x] = sum;
		__syncthreads();

		reduce<GpuGaussNewtonSolver::CTA_SIZE>(smem);

		if (threadIdx.x == 0)
			_g_common_q[blockIdx.x] = smem[0];
	}


	__global__ void factor_all_nodes_kernel(float4* nodesDqVw, int n, Tbx::Dual_quat_cu rigid_inv)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		if (i >= n)
			return;
		
		Tbx::Dual_quat_cu dq = rigid_inv * pack_dual_quat(nodesDqVw[3 * i], nodesDqVw[3 * i + 1]);
		unpack_dual_quat(dq, nodesDqVw[3 * i], nodesDqVw[3 * i + 1]);
	}

	// optional, factor out common rigid transformations among all nodes
	void GpuGaussNewtonSolver::factor_out_rigid()
	{
		if (m_pWarpField == nullptr)
			throw std::exception("GpuGaussNewtonSolver::solve: null pointer");
		if (m_pWarpField->getNumLevels() < 2)
			throw std::exception("non-supported levels of warp field!");
		if (m_pWarpField->getNumNodesInLevel(0) == 0)
		{
			printf("no warp nodes, return\n");
			return;
		}
		const int num0 = m_pWarpField->getNumNodesInLevel(0);
		const int numAll = m_pWarpField->getNumAllNodes();

		Tbx::Dual_quat_cu dq(Tbx::Quat_cu(0,0,0,0), Tbx::Quat_cu(0,0,0,0));
		hipMemcpyToSymbol(HIP_SYMBOL(_g_common_q), &dq, sizeof(Tbx::Dual_quat_cu));

		reduce_all_nodes_kernel << <8, GpuGaussNewtonSolver::CTA_SIZE >> >(
			m_pWarpField->getNodesDqVwPtr(0), num0);
		cudaSafeCall(hipGetLastError(), "reduce_all_nodes_kernel");
		hipMemcpyFromSymbol(&dq, HIP_SYMBOL(_g_common_q), sizeof(Tbx::Dual_quat_cu));

		if (dq.get_non_dual_part().norm() > Tbx::Dual_quat_cu::epsilon())
		{
			dq.normalize();
			m_pWarpField->set_rigidTransform(
				m_pWarpField->get_rigidTransform() * dq.to_transformation());

			for (int lv = 0; lv < m_pWarpField->getNumLevels(); lv++)
			{
				int numLv = m_pWarpField->getNumNodesInLevel(lv);
				if (numLv == 0)
					break;
				factor_all_nodes_kernel << <divUp(numLv, GpuGaussNewtonSolver::CTA_SIZE),
					GpuGaussNewtonSolver::CTA_SIZE >> >(m_pWarpField->getNodesDqVwPtr(lv), numLv, dq.conjugate());
			}
			cudaSafeCall(hipGetLastError(), "factor_all_nodes_kernel");

			// re-extract info
			m_pWarpField->extract_nodes_info_no_allocation(m_nodesKnn, m_twist, m_nodesVw);
			checkNan(m_twist, numAll * 6, "twist after factoring rigid");
		}
	}
#pragma endregion

}
#include "hip/hip_runtime.h"
#include "GpuKdTree.h"
#include "cudpp\thrust_wrapper.h"
#include "hip/hip_vector_types.h"
#include "cudpp\ModerGpuWrapper.h"
#include "GpuHeap.h"
#include "WarpField.h"

namespace dfusion
{
#define CHECK_ZERO(a){if(a)printf("!!!error: %s=%d\n", #a, a);}
	texture<int, hipTextureType1D, hipReadModeElementType> g_mempool_tex;
	texture<float4, hipTextureType1D, hipReadModeElementType> g_ele_low_high_tex;
	__constant__ int g_ele_low_high_tex_off_d[3];

#pragma region offsets
	__constant__ int g_mempool_tex_offs[26]; // size by int but not byte
	__device__ __forceinline__ int get_mempool_tex_ipoint_off(int i = 0){ return g_mempool_tex_offs[0 + i]; }
	__device__ __forceinline__ int get_mempool_tex_point_off(int i = 0){ return g_mempool_tex_offs[1 + i]; }
	__device__ __forceinline__ int get_mempool_tex_low_off(int i = 0){ return g_mempool_tex_offs[2 + i]; }
	__device__ __forceinline__ int get_mempool_tex_high_off(int i = 0){ return g_mempool_tex_offs[3 + i]; }
	__device__ __forceinline__ int get_mempool_tex_px_off(int i = 0){ return g_mempool_tex_offs[4 + i]; }
	__device__ __forceinline__ int get_mempool_tex_py_off(int i = 0){ return g_mempool_tex_offs[5 + i]; }
	__device__ __forceinline__ int get_mempool_tex_pz_off(int i = 0){ return g_mempool_tex_offs[6 + i]; }
	__device__ __forceinline__ int get_mempool_tex_tpx_off(int i = 0){ return g_mempool_tex_offs[7 + i]; }
	__device__ __forceinline__ int get_mempool_tex_tpy_off(int i = 0){ return g_mempool_tex_offs[8 + i]; }
	__device__ __forceinline__ int get_mempool_tex_tpz_off(int i = 0){ return g_mempool_tex_offs[9 + i]; }
	__device__ __forceinline__ int get_mempool_tex_split_off(int i = 0){ return g_mempool_tex_offs[10 + i]; }
	__device__ __forceinline__ int get_mempool_tex_child1_off(int i = 0){ return g_mempool_tex_offs[11 + i]; }
	__device__ __forceinline__ int get_mempool_tex_parent_off(int i = 0){ return g_mempool_tex_offs[12 + i]; }
	__device__ __forceinline__ int get_mempool_tex_idxx_off(int i = 0){ return g_mempool_tex_offs[13 + i]; }
	__device__ __forceinline__ int get_mempool_tex_idxy_off(int i = 0){ return g_mempool_tex_offs[14 + i]; }
	__device__ __forceinline__ int get_mempool_tex_idxz_off(int i = 0){ return g_mempool_tex_offs[15 + i]; }
	__device__ __forceinline__ int get_mempool_tex_ownerx_off(int i = 0){ return g_mempool_tex_offs[16 + i]; }
	__device__ __forceinline__ int get_mempool_tex_ownery_off(int i = 0){ return g_mempool_tex_offs[17 + i]; }
	__device__ __forceinline__ int get_mempool_tex_ownerz_off(int i = 0){ return g_mempool_tex_offs[18 + i]; }
	__device__ __forceinline__ int get_mempool_tex_lrx_off(int i = 0){ return g_mempool_tex_offs[19 + i]; }
	__device__ __forceinline__ int get_mempool_tex_lry_off(int i = 0){ return g_mempool_tex_offs[20 + i]; }
	__device__ __forceinline__ int get_mempool_tex_lrz_off(int i = 0){ return g_mempool_tex_offs[21 + i]; }
	__device__ __forceinline__ int get_mempool_tex_tidx_off(int i = 0){ return g_mempool_tex_offs[22 + i]; }
	__device__ __forceinline__ int get_mempool_tex_towner_off(int i = 0){ return g_mempool_tex_offs[23 + i]; }
	__device__ __forceinline__ int get_mempool_tex_tmisc_off(int i = 0){ return g_mempool_tex_offs[24 + i]; }
	__device__ __forceinline__ int get_mempool_tex_alloc_off(int i = 0){ return g_mempool_tex_offs[25 + i]; }
	__device__ __forceinline__ int get_ele_low_high_tex_low_off(int i=0){ return g_ele_low_high_tex_off_d[1 + i]; }
	__device__ __forceinline__ int get_ele_low_high_tex_high_off(int i = 0){ return g_ele_low_high_tex_off_d[2 + i]; }
#pragma endregion

	typedef GpuKdTree::SplitInfo SplitInfo;
	//! used to update the left/right pointers and aabb infos after the node splits
	struct SetLeftAndRightAndAABB
	{
		int maxPoints;
		int nElements;

		SplitInfo* nodes;
		int* counts;
		int* labels;
		float4* aabbMin;
		float4* aabbMax;
		const float* x, *y, *z;
		const int* ix, *iy, *iz;

		__host__ __device__ void operator()(int i)
		{
			int index = labels[i];
			int right = 0;
			int left = counts[i];
			nodes[index].left = left;
			if (i < nElements - 1) {
				right = counts[i + 1];
			}
			else { // index==nNodes
				right = maxPoints;
			}
			nodes[index].right = right;
			aabbMin[index].x = x[ix[left]];
			aabbMin[index].y = y[iy[left]];
			aabbMin[index].z = z[iz[left]];
			aabbMax[index].x = x[ix[right - 1]];
			aabbMax[index].y = y[iy[right - 1]];
			aabbMax[index].z = z[iz[right - 1]];
		}
	};

	//! computes the scatter target address for the split operation, see Sengupta,Harris,Zhang,Owen: Scan Primitives for GPU Computing
	//! in my use case, this is about 2x as fast as thrust::partition
	struct set_addr3
	{
		const int* val_, *f_;

		int npoints_;
		__device__ int operator()(int id)
		{
			int nf = f_[npoints_ - 1] + (val_[npoints_ - 1]);
			int f = f_[id];
			int t = id - f + nf;
			return val_[id] ? f : t;
		}
	};

	//! just for convenience: access a float4 by an index in [0,1,2]
	//! (casting it to a float* and accessing it by the index is way slower...)
	__host__ __device__ __forceinline__ float get_value_by_index(const float4& f, int i)
	{
		switch (i) {
		case 0:
			return f.x;
		case 1:
			return f.y;
		default:
			return f.z;
		}
	}

	__device__ __forceinline__ float read_ftex(int id, int offset)
	{
		int v = tex1Dfetch(g_mempool_tex, id + offset);
		return *((float*)&v);
	}
	__device__ __forceinline__ float4 read_f4tex(int id, int offset)
	{
		return make_float4(read_ftex(id << 2, offset), read_ftex((id << 2) + 1, offset),
			read_ftex((id << 2) + 2, offset), read_ftex((id << 2) + 3, offset));
	}
	__device__ __forceinline__ float4 read_f4tex_f4(int offset)
	{
		return tex1Dfetch(g_ele_low_high_tex, g_ele_low_high_tex_off_d[0] + offset);
	}
	__device__ __forceinline__ int read_itex(int id, int offset)
	{
		return tex1Dfetch(g_mempool_tex, id + offset);
	}
	__device__ __forceinline__ int2 read_i2tex(int id, int offset)
	{
		return make_int2(read_itex((id << 1), offset), read_itex((id << 1) + 1, offset));
	}

	//! - decide whether a node has to be split
	//! if yes:
	//! - allocate child nodes
	//! - set split axis as axis of maximum aabb length
	struct SplitNodes
	{
		typedef GpuKdTree::SplitInfo SplitInfo;
		int maxPointsPerNode;
		int* node_count;
		int* nodes_allocated;
		int* out_of_space;
		int* child1_;
		int* parent_;
		float4* aabbMin_;
		float4* aabbMax_;
		SplitInfo* splits_;

		// float4: aabbMin, aabbMax
		__device__ void operator()(int my_index) 
		{
			int child1 = child1_[my_index];
			SplitInfo s = splits_[my_index];
			float4 aabbMin = aabbMin_[my_index];
			float4 aabbMax = aabbMax_[my_index];

			bool split_node = false;
			// first, each thread block counts the number of nodes that it needs to allocate...
			__shared__ int block_nodes_to_allocate;
			if (threadIdx.x == 0) block_nodes_to_allocate = 0;
			__syncthreads();

			// don't split if all points are equal
			// (could lead to an infinite loop, and doesn't make any sense anyway)
			bool all_points_in_node_are_equal = aabbMin.x == aabbMax.x 
				&& aabbMin.y == aabbMax.y && aabbMin.z == aabbMax.z;

			int offset_to_global = 0;

			// maybe this could be replaced with a reduction...
			if ((child1 == -1) && (s.right - s.left > maxPointsPerNode) 
				&& !all_points_in_node_are_equal) { // leaf node
				split_node = true;
				offset_to_global = atomicAdd(&block_nodes_to_allocate, 2);
			}

			__syncthreads();
			__shared__ int block_left;
			__shared__ bool enough_space;
			// ... then the first thread tries to allocate this many nodes...
			if (threadIdx.x == 0) {
				block_left = atomicAdd(node_count, block_nodes_to_allocate);
				enough_space = block_left + block_nodes_to_allocate < *nodes_allocated;
				// if it doesn't succeed, no nodes will be created by this block
				if (!enough_space) {
					atomicAdd(node_count, -block_nodes_to_allocate);
					*out_of_space = 1;
				}
			}

			__syncthreads();
			// this thread needs to split it's node && there was enough space for all the nodes
			// in this block.
			//(The whole "allocate-per-block-thing" is much faster than letting each element allocate
			// its space on its own, because shared memory atomics are A LOT faster than
			// global mem atomics!)
			if (split_node && enough_space) {
				int left = block_left + offset_to_global;

				splits_[left] = s;
				splits_[left+1] = SplitInfo(make_int2(0,0));

				// split axis/position: middle of longest aabb extent
				float4 aabbDim = aabbMax - aabbMin;
				int maxDim = 0;
				float maxDimLength =  aabbDim.x;
				float4 splitVal = (aabbMax + aabbMin);
				splitVal *= 0.5f;
				for (int i = 1; i <= 2; i++) {
					float val = get_value_by_index(aabbDim, i);
					if (val > maxDimLength) {
						maxDim = i;
						maxDimLength = val;
					}
				}
				s.split_dim = maxDim;
				s.split_val = get_value_by_index(splitVal, maxDim);

				child1_[my_index] = left;
				splits_[my_index] = s;

				parent_[left] = my_index;
				parent_[left + 1] = my_index;
				child1_[left] = -1;
				child1_[left + 1] = -1;
			}
		}
	};

	//! mark a point as belonging to the left or right child of its current parent
	//! called after parents are split
	struct MovePointsToChildNodes
	{
		typedef GpuKdTree::SplitInfo SplitInfo;
		MovePointsToChildNodes(int* child1, SplitInfo* splits, 
			float* x, float* y, float* z, int* ox, int* oy, 
			int* oz, int* lrx, int* lry, int* lrz)
		: child1_(child1), splits_(splits), x_(x), y_(y), z_(z), ox_(ox), 
		oy_(oy), oz_(oz), lrx_(lrx), lry_(lry), lrz_(lrz){}

		//  int dim;
		//  float threshold;
		int* child1_;
		SplitInfo* splits_;

		// coordinate values
		float* x_, *y_, *z_;
		// owner indices -> which node does the point belong to?
		int* ox_, *oy_, *oz_;
		// temp info: will be set to 1 of a point is moved to the right child node, 0 otherwise
		// (used later in the scan op to separate the points of the children into continuous ranges)
		int* lrx_, *lry_, *lrz_;

		__device__ void operator()(int index, int point_ind1, int point_ind2, int point_ind3)
		{
			int owner = ox_[index]; 
			int leftChild = child1_[owner];
			int split_dim = 0;
			float dim_val1 = 0.f, dim_val2 = 0.f, dim_val3 = 0.f;
			SplitInfo split;
			lrx_[index] = 0;
			lry_[index] = 0;
			lrz_[index] = 0;
			// this element already belongs to a leaf node -> everything alright, no need to change anything
			if (leftChild == -1) 
				return;

			// otherwise: load split data, and assign this index to the new owner
			//split = splits_[owner];
			split = read_i2tex(owner, get_mempool_tex_split_off());
			split_dim = split.split_dim;

			switch (split_dim) {
			case 0:
				dim_val1 = x_[point_ind1];
				dim_val2 = x_[point_ind2];
				dim_val3 = x_[point_ind3];
				break;
			case 1:
				dim_val1 = y_[point_ind1];
				dim_val2 = y_[point_ind2];
				dim_val3 = y_[point_ind3];
				break;
			default:
				dim_val1 = z_[point_ind1];
				dim_val2 = z_[point_ind2];
				dim_val3 = z_[point_ind3];
				break;
			}

			ox_[index] = leftChild + (dim_val1 > split.split_val);
			oy_[index] = leftChild + (dim_val2 > split.split_val);
			oz_[index] = leftChild + (dim_val3 > split.split_val);
			lrx_[index] = (dim_val1 > split.split_val);
			lry_[index] = (dim_val2 > split.split_val);
			lrz_[index] = (dim_val3 > split.split_val);
		}
	};
	__global__ void splitNode_kernel(SplitNodes s, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < n)
			s(tid);
	}

	__global__ void movePointsToChildNodes_kernel(MovePointsToChildNodes s, 
		int* index_x, int* index_y, int* index_z, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < n)
		{
			s(tid, index_x[tid], index_y[tid], index_z[tid]);
		}
	}

	__global__ void for_each_SetLeftAndRightAndAABB_kernel(SetLeftAndRightAndAABB s, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < n)
			s(tid);
	}

	__global__ void collect_aabb_kernel(float4* aabb_min, float4* aabb_max,
		const float* x, const int* ix,
		const float* y, const int* iy,
		const float* z, const int* iz, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid == 0)
		{
			aabb_min[0] = make_float4(x[ix[0]], y[iy[0]], z[iz[0]], 0);
			aabb_max[0] = make_float4(x[ix[n-1]], y[iy[n-1]], z[iz[n-1]], 0);
		}
	}

	__global__ void set_addr3_kernel(set_addr3 sa, int* out, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < n)
		{
			out[tid] = sa(tid);
		}
	}

	template<class T>
	__global__ void resize_vec_kernel(const T* oldVec, T* newVec, int oldSize, int newSize, T val)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < newSize)
		{
			if (tid < oldSize)
				newVec[tid] = oldVec[tid];
			else
				newVec[tid] = val;
		}
	}

	__global__ void init_data_kernel(
		const float4* points_in, int stride_in_float4,
		float4* points_out, float* point_x, float* point_y, float* point_z,
		float* tmp_pt_x, float* tmp_pt_y, float* tmp_pt_z,
		int* index_x, int* index_y, int* index_z, int nPoints,
		int* child1, int* parent, SplitInfo* splits, int prealloc)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid < nPoints)
		{
			float4 xyz = points_in[tid*stride_in_float4];
			points_out[tid] = xyz;
			point_x[tid] = xyz.x;
			point_y[tid] = xyz.y;
			point_z[tid] = xyz.z;
			tmp_pt_x[tid] = xyz.x;
			tmp_pt_y[tid] = xyz.y;
			tmp_pt_z[tid] = xyz.z;
			index_x[tid] = tid;
			index_y[tid] = tid;
			index_z[tid] = tid;
		}
		if (tid < prealloc)
		{
			child1[tid] = -1;
			parent[tid] = -1;

			GpuKdTree::SplitInfo s;
			s.left = 0;
			s.right = 0;
			if (tid == 0)
				s.right = nPoints;
			splits[tid] = s;
		}
	}

	GpuKdTree::GpuKdTree()
	{
		nInputPoints_ = 0;
		nAllocatedPoints_ = 0;
		max_leaf_size_ = 0;
		prealloc_ = 0;

		// mempool_.ptr(), num = nInputPoints_
		input_points_ptr_ = nullptr;
		points_ptr_ = nullptr;
		aabb_min_ptr_ = nullptr;
		aabb_max_ptr_ = nullptr;
		points_x_ptr_ = nullptr;
		points_y_ptr_ = nullptr;
		points_z_ptr_ = nullptr;
		splits_ptr_ = nullptr;
		child1_ptr_ = nullptr;
		parent_ptr_ = nullptr;
		index_x_ptr_ = nullptr;
		index_y_ptr_ = nullptr;
		index_z_ptr_ = nullptr;
		owner_x_ptr_ = nullptr;
		owner_y_ptr_ = nullptr;
		owner_z_ptr_ = nullptr;
		leftright_x_ptr_ = nullptr;
		leftright_y_ptr_ = nullptr;
		leftright_z_ptr_ = nullptr;
		tmp_index_ptr_ = nullptr;
		tmp_owners_ptr_ = nullptr;
		tmp_misc_ptr_ = nullptr;
		allocation_info_ptr_ = nullptr;
	}

	void GpuKdTree::buildTree(const float4* points, int n, int stride_in_float4, int max_leaf_size)
	{
		if(n == 0)
			return;

		// memory allocation
		allocateMemPool(n, max_leaf_size);
		
		// data initialization
		// input_points
		{
			dim3 block(256);
			int num = max(nInputPoints_, prealloc_);
			dim3 grid(divUp(num, block.x));
			init_data_kernel << <grid, block >> >(points, stride_in_float4,
				input_points_ptr_, points_x_ptr_, points_y_ptr_,points_z_ptr_,
				tmp_pt_x_ptr_, tmp_pt_y_ptr_, tmp_pt_z_ptr_,
				index_x_ptr_, index_y_ptr_, index_z_ptr_, nInputPoints_,
				child1_ptr_, parent_ptr_, splits_ptr_, prealloc_);
			cudaSafeCall(hipGetLastError(), "init_data_kernel");
		}

		// allocation info
		cudaSafeCall(hipMemcpy(allocation_info_ptr_, allocation_info_host_.data(),
			allocation_info_host_.size()*sizeof(int), hipMemcpyHostToDevice),
			"GpuKdTree::buildTree:: memcpy");

		
		// create sorted index list -> can be used to compute AABBs in O(1)
		modergpu_wrapper::mergesort_by_key(tmp_pt_x_ptr_, index_x_ptr_, nInputPoints_);
		modergpu_wrapper::mergesort_by_key(tmp_pt_y_ptr_, index_y_ptr_, nInputPoints_);
		modergpu_wrapper::mergesort_by_key(tmp_pt_z_ptr_, index_z_ptr_, nInputPoints_);

		// bounding box info
		{
			dim3 block(1);
			dim3 grid(1);
			collect_aabb_kernel << <grid, block >> >(aabb_min_ptr_, aabb_max_ptr_,
				points_x_ptr_, index_x_ptr_, points_y_ptr_, index_y_ptr_,
				points_z_ptr_, index_z_ptr_, nInputPoints_);
		}
		
		int last_node_count = 0;
		for (int i = 0;; i++) 
		{
			SplitNodes sn;
			sn.maxPointsPerNode = max_leaf_size_;
			sn.node_count = allocation_info_ptr_ + NodeCount;
			sn.nodes_allocated = allocation_info_ptr_ + NodesAllocated;
			sn.out_of_space = allocation_info_ptr_ + OutOfSpace;
			sn.child1_ = child1_ptr_;
			sn.parent_ = parent_ptr_;
			sn.splits_ = splits_ptr_;
			sn.aabbMin_ = aabb_min_ptr_;
			sn.aabbMax_ = aabb_max_ptr_;
			if (last_node_count)
			{
				dim3 block(256);
				dim3 grid(divUp(last_node_count, block.x));
				splitNode_kernel << <grid, block >> >(sn, last_node_count);
			}

			// copy allocation info to host
			cudaSafeCall(hipMemcpy(allocation_info_host_.data(), allocation_info_ptr_,
				allocation_info_host_.size()*sizeof(int), hipMemcpyDeviceToHost),
				"GpuKdTree::buildTree, memcpy 1");

			if (last_node_count == allocation_info_host_[NodeCount]) // no more nodes were split -> done
				break;
			
			last_node_count = allocation_info_host_[NodeCount];

			// a node was un-splittable due to a lack of space
			if (allocation_info_host_[OutOfSpace] == 1) 
			{
				printf("GpuKdTree::buildTree(): warning: dynamic resize needed!\n");
				resize_node_vectors(allocation_info_host_[NodesAllocated] * 2);
				allocation_info_host_[OutOfSpace] = 0;
				allocation_info_host_[NodesAllocated] *= 2;
				cudaSafeCall(hipMemcpy(allocation_info_ptr_, allocation_info_host_.data(),
					allocation_info_host_.size()*sizeof(int), hipMemcpyHostToDevice),
					"GpuKdTree::buildTree, memcpy 2");
			}

			// foreach point: point was in node that was split?move it to child (leaf) node : do nothing
			MovePointsToChildNodes sno(child1_ptr_, splits_ptr_, points_x_ptr_,
				points_y_ptr_, points_z_ptr_, owner_x_ptr_, owner_y_ptr_,
				owner_z_ptr_, leftright_x_ptr_, leftright_y_ptr_, leftright_z_ptr_
				);
			{
				dim3 block(256);
				dim3 grid(divUp(nInputPoints_, block.x));
				movePointsToChildNodes_kernel << <grid, block >> >(sno, 
					index_x_ptr_, index_y_ptr_, index_z_ptr_, nInputPoints_);
			}

			// move points around so that each leaf node's points are continuous
			separate_left_and_right_children(index_x_ptr_, owner_x_ptr_, tmp_index_ptr_, 
				tmp_owners_ptr_, leftright_x_ptr_);
			hipMemcpy(index_x_ptr_, tmp_index_ptr_, nInputPoints_*sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(owner_x_ptr_, tmp_owners_ptr_, nInputPoints_*sizeof(int), hipMemcpyDeviceToDevice);
			separate_left_and_right_children(index_y_ptr_, owner_y_ptr_, tmp_index_ptr_, tmp_owners_ptr_,
				leftright_y_ptr_, false);
			hipMemcpy(index_y_ptr_, tmp_index_ptr_, nInputPoints_*sizeof(int), hipMemcpyDeviceToDevice);
			separate_left_and_right_children(index_z_ptr_, owner_z_ptr_, tmp_index_ptr_, tmp_owners_ptr_, 
				leftright_z_ptr_, false);
			hipMemcpy(index_z_ptr_, tmp_index_ptr_, nInputPoints_*sizeof(int), hipMemcpyDeviceToDevice);

			// calculate new AABB etc
			update_leftright_and_aabb(points_x_ptr_, points_y_ptr_, points_z_ptr_, index_x_ptr_,
				index_y_ptr_, index_z_ptr_, owner_x_ptr_, splits_ptr_, aabb_min_ptr_, aabb_max_ptr_);
		} 
		
		thrust_wrapper::gather(input_points_ptr_, index_x_ptr_, points_ptr_, nInputPoints_);

	}

	void GpuKdTree::allocateMemPool(int nInputPoints, int maxLeafSize)
	{
		nInputPoints_ = nInputPoints;
		max_leaf_size_ = maxLeafSize;
		if (nAllocatedPoints_ < nInputPoints_)
		{
			nAllocatedPoints_ = ceil(nInputPoints_ * 1.5);
			prealloc_ = divUp(nAllocatedPoints_ * 16, max_leaf_size_);
			mempool_.create(
				nAllocatedPoints_*sizeof(float4) * 2 +
				prealloc_ * sizeof(float4) * 2 +
				nAllocatedPoints_ * sizeof(float) * 6 +
				prealloc_ * sizeof(SplitInfo) +
				prealloc_ * sizeof(int) * 2 +
				nAllocatedPoints_ * sizeof(int) * 12 +
				4
				);
			printf("GpuKdTree: re-allocate %d\n", nInputPoints);

			// assigne buffers
			input_points_ptr_ = (float4*)mempool_.ptr();
			points_ptr_ = input_points_ptr_ + nAllocatedPoints_;
			aabb_min_ptr_ = points_ptr_ + nAllocatedPoints_;
			aabb_max_ptr_ = aabb_min_ptr_ + prealloc_;
			points_x_ptr_ = (float*)(aabb_max_ptr_ + prealloc_);
			points_y_ptr_ = points_x_ptr_ + nAllocatedPoints_;
			points_z_ptr_ = points_y_ptr_ + nAllocatedPoints_;
			tmp_pt_x_ptr_ = points_z_ptr_ + nAllocatedPoints_;
			tmp_pt_y_ptr_ = tmp_pt_x_ptr_ + nAllocatedPoints_;
			tmp_pt_z_ptr_ = tmp_pt_y_ptr_ + nAllocatedPoints_;
			splits_ptr_ = (SplitInfo*)(tmp_pt_z_ptr_ + nAllocatedPoints_);
			child1_ptr_ = (int*)(splits_ptr_+prealloc_);
			parent_ptr_ = child1_ptr_ + prealloc_;
			index_x_ptr_ = parent_ptr_ + prealloc_;
			index_y_ptr_ = index_x_ptr_ + nAllocatedPoints_;
			index_z_ptr_ = index_y_ptr_ + nAllocatedPoints_;
			owner_x_ptr_ = index_z_ptr_ + nAllocatedPoints_;
			owner_y_ptr_ = owner_x_ptr_ + nAllocatedPoints_;
			owner_z_ptr_ = owner_y_ptr_ + nAllocatedPoints_;
			leftright_x_ptr_ = owner_z_ptr_ + nAllocatedPoints_;
			leftright_y_ptr_ = leftright_x_ptr_ + nAllocatedPoints_;
			leftright_z_ptr_ = leftright_y_ptr_ + nAllocatedPoints_;
			tmp_index_ptr_ = leftright_z_ptr_ + nAllocatedPoints_;
			tmp_owners_ptr_ = tmp_index_ptr_ + nAllocatedPoints_;
			tmp_misc_ptr_ = tmp_owners_ptr_ + nAllocatedPoints_;
			allocation_info_ptr_ = tmp_misc_ptr_ + nAllocatedPoints_;
		}


		allocation_info_host_.resize(3);
		allocation_info_host_[GpuKdTree::NodeCount] = 1;
		allocation_info_host_[GpuKdTree::NodesAllocated] = prealloc_;
		allocation_info_host_[GpuKdTree::OutOfSpace] = 0;

		// reset mem
		hipMemset(mempool_.ptr(), 0, mempool_.size()*mempool_.elem_size);

		// bind src to texture
		bindTextures();
	}

	namespace KdTreeCudaPrivate
	{	
		//! implementation of L2 distance for the CUDA kernels
		struct CudaL2Distance
		{
			static float __host__ __device__ __forceinline__ axisDist(float a, float b)
			{
				return (a - b)*(a - b);
			}

			static float __host__ __device__ __forceinline__ dist(float4 a, float4 b)
			{
				return (a.x - b.x)*(a.x - b.x) + (a.y - b.y)*(a.y - b.y) + (a.z - b.z)*(a.z - b.z);
			}
		};

		//! result set for the 1nn search. Doesn't do any global memory accesses on its own,
		template< typename DistanceType, typename IndexType = int >
		struct SingleResultSet
		{
			IndexType bestIndex;
			DistanceType bestDist;
			enum{ ResultK = 1 };

			__device__ __host__ SingleResultSet() : 
				bestIndex(IndexType(-1)), bestDist(INFINITY),
				resultDist(nullptr), resultIndex(nullptr),
				selfIndex(IndexType(-1)){ }

			__device__ inline float worstDist()
			{
				return bestDist;
			}

			__device__ inline void insert(int index, DistanceType dist)
			{
				if (dist <= bestDist && index != selfIndex) {
					bestIndex = index;
					bestDist = dist;
				}
			}

			DistanceType* resultDist;
			IndexType* resultIndex;
			IndexType selfIndex;

			__device__ inline void setResultLocation(DistanceType* dists, 
				IndexType* index, int thread, int knnStride, bool excludeSelf)
			{
				if (dists)
					resultDist = dists + thread * knnStride;
				if (index)
					resultIndex = index + thread * knnStride;
				if (excludeSelf)
					selfIndex = thread;
			}

			__device__ inline void finish()
			{
				if (resultDist)
					resultDist[0] = bestDist;
				if (resultIndex)
					resultIndex[0] = bestIndex;
			}
		};

		template< typename DistanceType >
		struct GreaterThan
		{
			__device__
			bool operator()(DistanceType a, DistanceType b)
			{
				return a>b;
			}
		};

		// using this and the template uses 2 or 3 registers more than the direct implementation in the kNearestKernel, but
		// there is no speed difference.
		// Setting useHeap as a template parameter leads to a whole lot of things being
		// optimized away by nvcc.
		// Register counts are the same as when removing not-needed variables in explicit specializations
		// and the "if( useHeap )" branches are eliminated at compile time.
		// The downside of this: a bit more complex kernel launch code.
		template< typename DistanceType, int K, typename IndexType = int>
		struct KnnResultSet
		{
			int foundNeighbors;
			DistanceType largestHeapDist;
			IndexType maxDistIndex;
			const bool sorted;
			enum{ ResultK = K };

			__device__ __host__ KnnResultSet(bool sortResults) : 
				foundNeighbors(0), largestHeapDist(INFINITY), sorted(sortResults),
				resultIndex_out(0), resultDist_out(0), selfIndex(IndexType(-1)){
				for (int i = 0; i < K; i++)
				{
					resultDist[i] = INFINITY;
					resultIndex[i] = IndexType(-1);
				}
			}

			__device__ inline DistanceType worstDist()
			{
				return largestHeapDist;
			}

			__device__ inline void insert(IndexType index, DistanceType dist)
			{
				if (index == selfIndex)
					return;
				if (foundNeighbors < K) {
					resultDist[foundNeighbors] = dist;
					resultIndex[foundNeighbors] = index;
					if (foundNeighbors == K - 1)
						findLargestDistIndex();
					foundNeighbors++;
				}
				else if (dist < largestHeapDist) {
					resultDist[maxDistIndex] = dist;
					resultIndex[maxDistIndex] = index;
					findLargestDistIndex();
				}
			}

			__device__ void findLargestDistIndex()
			{
				largestHeapDist = resultDist[0];
				maxDistIndex = 0;
				for (int i = 1; i<K; i++)
				if (resultDist[i] > largestHeapDist) {
					maxDistIndex = i;
					largestHeapDist = resultDist[i];
				}
			}

			float resultDist[K];
			IndexType resultIndex[K];
			float* resultDist_out;
			IndexType* resultIndex_out;
			IndexType selfIndex;

			__device__ inline void setResultLocation(DistanceType* dists, IndexType* index, 
				int thread, int knnStride, bool excludeSelf)
			{
				if (index)
					resultIndex_out = index + thread*knnStride;
				if (dists)
					resultDist_out = dists + thread*knnStride;
				for (int i = 0; i < K; i++) {
					resultDist[i] = INFINITY;
					resultIndex[i] = IndexType(-1);
				}
				if (excludeSelf)
					selfIndex = thread;
			}

			__host__ __device__ inline void finish()
			{
				if (sorted) {
					flann::cuda::heap::make_heap(resultDist, resultIndex, K, GreaterThan<DistanceType>());
					for (int i = K - 1; i>0; i--) {
						flann::cuda::swap(resultDist[0], resultDist[i]);
						flann::cuda::swap(resultIndex[0], resultIndex[i]);
						flann::cuda::heap::sift_down(resultDist, resultIndex, 0, i, GreaterThan<DistanceType>());
					}
				}

				if (resultDist_out)
				{
					for (int i = 0; i < K; i++)
						resultDist_out[i] = resultDist[i];
				}
				if (resultIndex_out)
				{
					for (int i = 0; i < K; i++)
						resultIndex_out[i] = resultIndex[i];
				}
			}
		};

		template< typename GPUResultSet>
		__device__ void searchNeighbors(const float4& q, GPUResultSet& result)
		{
			bool backtrack = false;
			int lastNode = -1;
			int current = 0;

			GpuKdTree::SplitInfo split;
			while (true) {
				if (current == -1) break;
				split = read_i2tex(current, get_mempool_tex_split_off());

				float diff1 = (split.split_dim == 0)*(q.x - split.split_val)
					+ (split.split_dim == 1)*(q.y - split.split_val)
					+ (split.split_dim == 2)*(q.z - split.split_val);

				// children are next to each other: leftChild+1 == rightChild
				int leftChild = read_itex(current, get_mempool_tex_child1_off());
				int bestChild = leftChild +(diff1 >= 0);
				int otherChild = leftChild +(diff1 < 0);

				if (!backtrack) {
					/* If this is a leaf node, then do check and return. */
					if (leftChild == -1) {
						for (int i = split.left; i < split.right; ++i) {
							float dist = CudaL2Distance::dist(read_f4tex_f4(i), q);
							result.insert(read_itex(i, get_mempool_tex_idxx_off()), dist);
						}

						backtrack = true;
						lastNode = current;
						current = read_itex(current, get_mempool_tex_parent_off());
					}
					else { // go to closer child node
						lastNode = current;
						current = bestChild;
					}
				}
				else { 
					// continue moving back up the tree or visit far node?
					// minimum possible distance between query point and a point inside the AABB
					float4 aabbMin = read_f4tex_f4(otherChild + get_ele_low_high_tex_low_off());
					float4 aabbMax = read_f4tex_f4(otherChild + get_ele_low_high_tex_high_off());
					float mindistsq = (q.x < aabbMin.x) * CudaL2Distance::axisDist(q.x, aabbMin.x)
						+ (q.x > aabbMax.x) * CudaL2Distance::axisDist(q.x, aabbMax.x)
						+ (q.y < aabbMin.y) * CudaL2Distance::axisDist(q.y, aabbMin.y)
						+ (q.y > aabbMax.y) * CudaL2Distance::axisDist(q.y, aabbMax.y)
						+ (q.z < aabbMin.z) * CudaL2Distance::axisDist(q.z, aabbMin.z)
						+ (q.z > aabbMax.z) * CudaL2Distance::axisDist(q.z, aabbMax.z);

					//  the far node was NOT the last node (== not visited yet) 
					//  AND there could be a closer point in it
					if ((lastNode == bestChild) && (mindistsq <= result.worstDist())) 
					{
						lastNode = current;
						current = otherChild;
						backtrack = false;
					}
					else {
						lastNode = current;
						current = read_itex(current, get_mempool_tex_parent_off());
					}
				}
			}
		}

		template< typename GPUResultSet, typename IndexType = int>
		__global__ void nearestKernel(const float4* query,
			int query_stride_in_float4,
			IndexType* resultIndex, float* resultDist,
			int querysize, GPUResultSet result, int knnStride,
			bool excludeSelf = false
			)
		{
			typedef float DistanceType;
			typedef float ElementType;
			//                  typedef DistanceType float;
			int tid = blockDim.x*blockIdx.x + threadIdx.x;

			if (tid >= querysize) return;

			float4 q = query[tid*query_stride_in_float4];

			result.setResultLocation(resultDist, resultIndex, tid, knnStride, excludeSelf);
			searchNeighbors(q, result);
			result.finish();
		}

		template< typename GPUResultSet>
		__global__ void nearestKernel(hipSurfaceObject_t volumeSurf, int3 begin, int3 end,
			float3 origion, float voxelSize, GPUResultSet result, int knnStride, bool excludeSelf=false
			)
		{
			typedef float DistanceType;
			typedef float ElementType;

			int ix = blockDim.x*blockIdx.x + threadIdx.x + begin.x;
			int iy = blockDim.y*blockIdx.y + threadIdx.y + begin.y;
			int iz = blockDim.z*blockIdx.z + threadIdx.z + begin.z;

			if (ix < end.x && iy < end.y && iz < end.z)
			{
				float4 q;
				q.x = origion.x + ix*voxelSize;
				q.y = origion.y + iy*voxelSize;
				q.z = origion.z + iz*voxelSize;
				q.w = 0.f;
				searchNeighbors(q, result);

				result.setResultLocation(nullptr, nullptr, 0, knnStride, excludeSelf);
				searchNeighbors(q, result);
				result.finish();

				KnnIdx knn = make_knn(KnnIdxType(-1));
				for (int k = 0; k < GPUResultSet::ResultK; k++)
					knn_k(knn, k) = result.resultIndex[k];
				write_knn(knn, volumeSurf, ix, iy, iz);
			}
		}
	}

	void GpuKdTree::knnSearchGpu(const float4* queries, int query_stride_in_float4,
		ushort* indices, float* dists, size_t knn, size_t n, size_t knnStride, bool excludeSelf) const
	{
		if (n == 0)
			return;
		int threadsPerBlock = 256;
		int blocksPerGrid = divUp(n, threadsPerBlock);
		bool sorted = true;

		// bind src to texture
		bindTextures();

		switch (knn)
		{
		case 1:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::SingleResultSet<float, ushort>(), 
				knnStride, 
				excludeSelf
				);
			break;
		case 2:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 2, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 3:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 3, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 4:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 4, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 5:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 5, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 6:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 6, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 7:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 7, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 8:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 8, ushort>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		default:
			throw std::exception("non-supported K in KnnSearch!");
		}
	}

	void GpuKdTree::knnSearchGpu(const float4* queries, int query_stride_in_float4, 
		int* indices, float* dists, size_t knn, size_t n, size_t knnStride, bool excludeSelf) const
	{
		if (n == 0)
			return;
		int threadsPerBlock = 256;
		int blocksPerGrid = divUp(n, threadsPerBlock);
		bool sorted = true;

		// bind src to texture
		bindTextures();

		switch (knn)
		{
		case 1:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::SingleResultSet<float>(),
				knnStride,
				excludeSelf
				);
			break;
		case 2:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 2>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 3:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 3>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 4:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 4>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 5:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 5>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 6:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 6>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 7:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 7>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		case 8:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				queries,
				query_stride_in_float4,
				indices,
				dists,
				n,
				KdTreeCudaPrivate::KnnResultSet<float, 8>(sorted),
				knnStride,
				excludeSelf
				);
			break;
		default:
			throw std::exception("non-supported K in KnnSearch!");
		}
	}

	void GpuKdTree::knnSearchGpu(hipSurfaceObject_t volumeSurf, int3 begin, int3 end,
		float3 origion, float voxelSize, size_t knn) const
	{
		if (begin.x >= end.x || begin.y >= end.y || begin.z >= end.z)
			return;

		// bind src to texture
		bindTextures();

		dim3 threadsPerBlock(32, 8, 2);
		dim3 blocksPerGrid(divUp(end.x-begin.x, threadsPerBlock.x),
			divUp(end.y - begin.y, threadsPerBlock.y),
			divUp(end.z - begin.z, threadsPerBlock.z));
		bool sorted = true;

		switch (knn)
		{
		case 0:
			break;
		case 1:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 1, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 2:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 2, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 3:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 3, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 4:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 4, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 5:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 5, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 6:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 6, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 7:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 7, KnnIdxType>(sorted),
				KnnK
				);
			break;
		case 8:
			KdTreeCudaPrivate::nearestKernel << <blocksPerGrid, threadsPerBlock >> > (
				volumeSurf,
				begin,
				end,
				origion,
				voxelSize,
				KdTreeCudaPrivate::KnnResultSet<float, 8, KnnIdxType>(sorted),
				KnnK
				);
			break;
		default:
			throw std::exception("non-supported knnK!");
		}
	}

	void GpuKdTree::update_leftright_and_aabb(
		const float* x,
		const float* y,
		const float* z,
		const int* ix,
		const int* iy,
		const int* iz,
		const int* owners,
		SplitInfo* splits,
		float4* aabbMin,
		float4* aabbMax)
	{
		int* labelsUnique = tmp_owners_ptr_;
		int* countsUnique = tmp_index_ptr_;
		// assume: points of each node are continuous in the array

		// find which nodes are here, and where each node's points begin and end
		int unique_labels = thrust_wrapper::unique_counting_by_key_copy(
			owners, 0, labelsUnique, countsUnique, nInputPoints_);

		// update the info
		SetLeftAndRightAndAABB s;
		s.maxPoints = nInputPoints_;
		s.nElements = unique_labels;
		s.nodes = splits;
		s.counts = countsUnique;
		s.labels = labelsUnique;
		s.x = x;
		s.y = y;
		s.z = z;
		s.ix = ix;
		s.iy = iy;
		s.iz = iz;
		s.aabbMin = aabbMin;
		s.aabbMax = aabbMax;

		dim3 block(256);
		dim3 grid(divUp(unique_labels, block.x));
		for_each_SetLeftAndRightAndAABB_kernel << <grid, block >> >(s, unique_labels);
		cudaSafeCall(hipGetLastError(), "GpuKdTree::update_leftright_and_aabb");
	}

	//! Separates the left and right children of each node into continuous parts of the array.
	//! More specifically, it seperates children with even and odd node indices because nodes are always
	//! allocated in pairs -> child1==child2+1 -> child1 even and child2 odd, or vice-versa.
	//! Since the split operation is stable, this results in continuous partitions
	//! for all the single nodes.
	//! (basically the split primitive according to sengupta et al)
	//! about twice as fast as thrust::partition
	void GpuKdTree::separate_left_and_right_children(
		int* key_in,
		int* val_in,
		int* key_out,
		int* val_out,
		int* left_right_marks,
		bool scatter_val_out)
	{
		int* f_tmp = val_out;
		int* addr_tmp = tmp_misc_ptr_;

		thrust_wrapper::exclusive_scan(left_right_marks, f_tmp, nInputPoints_);

		set_addr3 sa;
		sa.val_ = left_right_marks;
		sa.f_ = f_tmp;
		sa.npoints_ = nInputPoints_;
		{
			dim3 block(256);
			dim3 grid(divUp(nInputPoints_, block.x));
			set_addr3_kernel << <grid, block >> >(sa, addr_tmp, nInputPoints_);
			cudaSafeCall(hipGetLastError(), "set_addr3_kernel");
		}
		thrust_wrapper::scatter(key_in, addr_tmp, key_out, nInputPoints_);
		if (scatter_val_out) 
			thrust_wrapper::scatter(val_in, addr_tmp, val_out, nInputPoints_);
	}

	template<class T>
	static void resize_vec(DeviceArray<T>& oldVec, int new_size, T val)
	{
		DeviceArray<T> newVec;
		newVec.create(new_size);

		dim3 block(256);
		dim3 grid(divUp(new_size, block.x));
		resize_vec_kernel<<<grid, block>>>(oldVec.ptr(), newVec.ptr(), oldVec.size(), newVec.size(), val);
	}

	//! allocates additional space in all the node-related vectors.
	//! new_size elements will be added to all vectors.
	void GpuKdTree::resize_node_vectors(size_t new_size)
	{
		throw std::exception("not supported!");
		//resize_vec(child1_, new_size, -1);
		//resize_vec(parent_, new_size, -1);
		//SplitInfo s;
		//s.left = 0;
		//s.right = 0;
		//resize_vec(splits_, new_size, s);
		//float4 f = make_float4(0,0,0,0);
		//resize_vec(aabb_min_, new_size, f);
		//resize_vec(aabb_max_, new_size, f);
	}

	void GpuKdTree::bindTextures()const
	{
		size_t offset;
		hipChannelFormatDesc desc_int = hipCreateChannelDesc<int>();
		hipBindTexture(&offset, &g_mempool_tex, mempool_.ptr(), &desc_int,
			mempool_.size()*sizeof(int));
		CHECK_ZERO(offset);
		hipChannelFormatDesc desc_f4 = hipCreateChannelDesc<float4>();
		cudaSafeCall(hipBindTexture(&offset, &g_ele_low_high_tex, points_ptr_, &desc_f4,
			aabb_max_offset_byte() - points_offset_byte() + prealloc_*sizeof(float4)),
			"GpuKdTree::bindTextures 1");


		int offset_f4[3] = {
			int(offset / sizeof(float4)),
			int((aabb_min_offset_byte() - points_offset_byte()) / sizeof(float4)),
			int((aabb_max_offset_byte() - points_offset_byte()) / sizeof(float4))
		};
		cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_ele_low_high_tex_off_d), offset_f4, sizeof(offset_f4)),
			"GpuKdTree::bindTextures 2");

		// store texture offsets
		int offsets_tex[26] = {
			input_points_offset_byte() / 4,
			points_offset_byte() / 4,
			aabb_min_offset_byte() / 4,
			aabb_max_offset_byte() / 4,
			points_x_offset_byte() / 4,
			points_y_offset_byte() / 4,
			points_z_offset_byte() / 4,
			tpt_x_offset_byte() / 4,
			tpt_y_offset_byte() / 4,
			tpt_z_offset_byte() / 4,
			splits_offset_byte() / 4,
			child1_offset_byte() / 4,
			parent_offset_byte() / 4,
			index_x_offset_byte() / 4,
			index_y_offset_byte() / 4,
			index_z_offset_byte() / 4,
			owner_x_offset_byte() / 4,
			owner_y_offset_byte() / 4,
			owner_z_offset_byte() / 4,
			leftright_x_offset_byte() / 4,
			leftright_y_offset_byte() / 4,
			leftright_z_offset_byte() / 4,
			tmp_index_offset_byte() / 4,
			tmp_owners_offset_byte() / 4,
			tmp_misc_offset_byte() / 4,
			allocation_info_offset_byte() / 4
		};
		cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_mempool_tex_offs), offsets_tex, sizeof(offsets_tex)),
			"GpuKdTree::bindTextures, 3");
	}
}
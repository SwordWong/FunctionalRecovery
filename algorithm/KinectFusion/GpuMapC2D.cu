#include "hip/hip_runtime.h"
#include "GpuMapC2D.h"
#include "device_utils.h"
#include <hip/hip_vector_types.h>
#include ""
#include <thrust/sort.h>
#include <thrust/functional.h>


struct C2DMapper
{
	//const int* handPixel;
	const NUI_COLOR_IMAGE_POINT* coordMapping;
	uchar4* color_list;
	int* handflag;

	uchar4* color_src;
	uchar4* color_dst;

	dfusion::depthtype* depth;

	int color_dis_sq;

	int rows;
	int cols;

	int color_list_size;

	int num_hand_pixel;
	__device__ __forceinline__ void operator()(int row, int col)
	{
		if (row >= 0 && row < rows && col >= 0 && col < cols)
		{
			int index = row*cols + col;
			if (depth[index] == 0)
				return;
			int colorInDepthX = coordMapping[index].x;
			int colorInDepthY = coordMapping[index].y;

			unsigned int sourceColorIndex = colorInDepthX + (colorInDepthY * cols);
			uchar4 color = color_src[sourceColorIndex];

			color_dst[index] = color;
			if (toRemove(color))
			{
				//num_hand_pixel++;
				Remove(index);
			}
		}

	}
	__device__ __forceinline__ bool Remove(int index)
	{
		//if (row >= 0 && row < rows && col >= 0 && col < cols)
		//{
			depth[index] = 0;
			//int index = row*cols + col;
			handflag[index] = 1;
		//}
	}
	__device__ __forceinline__ bool toRemove(uchar4 color)
	{

		for (int i = 0; i < color_list_size; i++)
		{
			int dis_sq = 0;
			dis_sq += (color.x - color_list[i].x)*(color.x - color_list[i].x);
			dis_sq += (color.y - color_list[i].y)*(color.y - color_list[i].y);
			dis_sq += (color.z - color_list[i].z)*(color.z - color_list[i].z);
			if (dis_sq < color_dis_sq)
				return true;
		}
		return false;
	}

};

__global__ void Color2Depth_kernal(C2DMapper mapper)
{
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	mapper(row, col);
}
__global__ void init_index_kernal(int* index_d, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size)
		index_d[index] = index;
}
//void mapC2D_GPU(dfusion::depthtype* depth, unsigned char* dst_pBGRA,
//	const unsigned char* src_pBGRA, const std::vector<dfusion::PixelRGBA> &colorList, std::set<PixelPos> &handPixel,
//	std::vector<NUI_COLOR_IMAGE_POINT> &coordMapping, int color_dis)
//{
//	//printf("mapC2D_GPU:color_dis = %d\n", color_dis);
//	dfusion::depthtype* depth_d;
//	dfusion::PixelRGBA* src_color_d;
//	dfusion::PixelRGBA* dst_color_d;
//
//
//	NUI_COLOR_IMAGE_POINT* coordMapping_d;
//	dfusion::PixelRGBA* colorList_d;
//	bool* handflag_d;
//
//
//	hipMalloc((void**)&depth_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype));
//	hipMalloc((void**)&dst_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA));
//	hipMalloc((void**)&src_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA));
//	
//
//	hipMalloc((void**)&handflag_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(bool));
//	hipMalloc((void**)&coordMapping_d, coordMapping.size() * sizeof(NUI_COLOR_IMAGE_POINT));
//	hipMalloc((void**)&colorList_d, colorList.size() * sizeof(dfusion::PixelRGBA));
//
//	//printf("mapC2D_GPU:hipMalloc\n");
//
//
//	hipMemset(handflag_d, false, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT);
//	//hipMemcpy(handflag_d, handPixel.data(), handPixel.size() * sizeof(PixelPos), hipMemcpyHostToDevice);
//	hipMemcpy(coordMapping_d, coordMapping.data(), coordMapping.size() * sizeof(NUI_COLOR_IMAGE_POINT), hipMemcpyHostToDevice);
//	hipMemcpy(colorList_d, colorList.data(), colorList.size() * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);
//
//
//	hipMemcpy(depth_d, depth, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype), hipMemcpyHostToDevice);
//	hipMemcpy(src_color_d, src_pBGRA, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);
//	//hipMemcpy(dst_color_d, src_pBGRA, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);
//	//printf("mapC2D_GPU:hipMemcpy\n");
//
//	C2DMapper mapper;
//
//	mapper.depth = depth_d;
//	mapper.color_src = (uchar4*)src_color_d;
//	mapper.color_dst = (uchar4*)dst_color_d;
//	mapper.color_list = (uchar4*)colorList_d;
//	mapper.handflag = handflag_d;
//	mapper.coordMapping = coordMapping_d;
//	
//	mapper.color_list_size = colorList.size();
//	mapper.rows = dfusion::KINECT_HEIGHT;
//	mapper.cols = dfusion::KINECT_WIDTH;
//
//	mapper.color_dis_sq = color_dis*color_dis;
//	mapper.num_hand_pixel = 0;
//	dim3 block(32, 8);
//	dim3 grid(1, 1, 1);
//	grid.x = divUp(mapper.cols, block.x);
//	grid.y = divUp(mapper.rows, block.y);
//
//	//printf("mapC2D_GPU:init finished\n");
//
//	Color2Depth_kernal << <grid, block >> > (mapper);
//	cudaSafeCall(hipGetLastError(), "mapColor2Depth_GPU");
//
//	hipMemcpy(depth, depth_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype), hipMemcpyDeviceToHost);
//	hipMemcpy((dfusion::PixelRGBA*)dst_pBGRA, dst_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyDeviceToHost);
//	
//	
//	handPixel.clear();
//	//std::vector<bool> handflag;
//	bool handflag[dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT];
//	//printf("mapC2D_GPU:handflag.resize\n");
//	hipMemcpy(handflag, handflag_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(bool), hipMemcpyDeviceToHost);
//	//cudaSafeCall(hipGetLastError(), "mapColor2Depth_GPU");
//	//printf("mapC2D_GPU:hipMemcpy handflag\n");
//	for (int i = 0; i < dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT; i++)
//	{
//		//printf("mapC2D_GPU:i = %d\n",i);
//		if(handflag[i])
//			handPixel.insert(i);
//	}
//	//printf("mapC2D_GPU:handPixel size = %d\n",handPixel.size());
//
//	hipFree(depth_d);
//	hipFree(src_color_d);
//	hipFree(dst_color_d);
//
//	hipFree(colorList_d);
//	hipFree(handflag_d);
//	hipFree(coordMapping_d);
//}

GpuMapper::GpuMapper()
{
	printf("init GpuMapper\n");
	color_list_size = 0;

	hipMalloc((void**)&depth_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype));
	hipMalloc((void**)&dst_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA));
	hipMalloc((void**)&src_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA));


	hipMalloc((void**)&handflag_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(int));
	hipMalloc((void**)&handflag_index_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(int));
	hipMalloc((void**)&coordMapping_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(NUI_COLOR_IMAGE_POINT));
	//hipMalloc((void**)&colorList_d, colorList.size() * sizeof(dfusion::PixelRGBA));
	printf("GpuMapper init\n");
}

GpuMapper::~GpuMapper()
{
	hipFree(depth_d);
	hipFree(src_color_d);
	hipFree(dst_color_d);

	//hipFree(colorList_d);
	hipFree(handflag_d);
	hipFree(handflag_index_d);
	hipFree(coordMapping_d);
}

void GpuMapper::map(dfusion::depthtype * depth, unsigned char * dst_pBGRA, const unsigned char * src_pBGRA, const std::vector<dfusion::PixelRGBA>& colorList, std::set<PixelPos>* handPixel, std::vector<NUI_COLOR_IMAGE_POINT>& coordMapping, int color_dis)
{
	//printf("GpuMapper::map\n");
	hipMalloc((void**)&colorList_d, colorList.size() * sizeof(dfusion::PixelRGBA));
	
	hipMemset(handflag_d, 0, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT*sizeof(int));
	hipMemset(dst_color_d, 0, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA));
	
	hipMemcpy(coordMapping_d, coordMapping.data(), coordMapping.size() * sizeof(NUI_COLOR_IMAGE_POINT), hipMemcpyHostToDevice);
	hipMemcpy(colorList_d, colorList.data(), colorList.size() * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);


	hipMemcpy(depth_d, depth, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype), hipMemcpyHostToDevice);
	hipMemcpy(src_color_d, src_pBGRA, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);
	//hipMemcpy(dst_color_d, src_pBGRA, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);
	//printf("mapC2D_GPU:hipMemcpy\n");

	C2DMapper mapper;

	mapper.depth = depth_d;
	mapper.color_src = (uchar4*)src_color_d;
	mapper.color_dst = (uchar4*)dst_color_d;
	mapper.color_list = (uchar4*)colorList_d;
	mapper.handflag = handflag_d;
	mapper.coordMapping = coordMapping_d;

	color_list_size = colorList.size();

	mapper.color_list_size = colorList.size();
	mapper.rows = dfusion::KINECT_HEIGHT;
	mapper.cols = dfusion::KINECT_WIDTH;

	mapper.num_hand_pixel = 0;

	mapper.color_dis_sq = color_dis*color_dis;

	dim3 block(32, 8);
	dim3 grid(1, 1, 1);
	grid.x = divUp(mapper.cols, block.x);
	grid.y = divUp(mapper.rows, block.y);

	//printf("mapC2D_GPU:init finished\n");

	Color2Depth_kernal << <grid, block >> > (mapper);
	//cudaSafeCall(hipGetLastError(), "mapColor2Depth_GPU");

	hipMemcpy(depth, depth_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::depthtype), hipMemcpyDeviceToHost);
	hipMemcpy((dfusion::PixelRGBA*)dst_pBGRA, dst_color_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(dfusion::PixelRGBA), hipMemcpyDeviceToHost);

	if (handPixel)
	{
		handPixel->clear();
		//std::vector<bool> handflag;

		//printf("mapC2D_GPU:handflag.resize\n");

		if (color_list_size > 0)
		{
			int n = dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT;
			//printf("mapper.num_hand_pixel = %d\n", mapper.num_hand_pixel);

			//init_handflag_index();

			//int* handflag = new int[dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT];
			//int* handflag_index = new int[dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT];

			for (int i = 0; i < n; i++)handflag_index_h[i] = i;

			hipMemcpy(handflag_h, handflag_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
			//hipMemcpy(handflag_index, handflag_index_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
			//
			thrust::sort_by_key(handflag_h, handflag_h + n, handflag_index_h);
			//printf("sorted\n");
			int index = n - 1;
			while (index >= 0 && handflag_h[index])
			{
				handPixel->insert(handflag_index_h[index]);
				index--;
			}
			/*for (int i = 0; i < n; i++)
			{
			if (handflag[i]);
			handPixel.insert(i);
			}*/
			//delete[] handflag_index;
			//delete[] handflag;
		}

		//printf("mapC2D_GPU:handPixel size = %d\n", handPixel->size());
	}
	

	
	hipFree(colorList_d);
	
}

void GpuMapper::init_handflag_index()
{
	dim3 block(256, 1);
	dim3 grid(1, 1, 1);
	grid.x = divUp(dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT, block.x);
	init_index_kernal << <grid, block >> > (handflag_index_d, dfusion::KINECT_WIDTH * dfusion::KINECT_HEIGHT);
}

void GpuMapper::set_color_list(const std::vector<dfusion::PixelRGBA>& colorList)
{
	release_color_list();
	hipMalloc((void**)&colorList_d, colorList.size() * sizeof(dfusion::PixelRGBA));
	hipMemcpy(colorList_d, colorList.data(), colorList.size() * sizeof(dfusion::PixelRGBA), hipMemcpyHostToDevice);

	color_list_size = colorList.size();
}

void GpuMapper::release_color_list()
{
	if(color_list_size > 0)
		hipFree(colorList_d);
	color_list_size = 0;
}

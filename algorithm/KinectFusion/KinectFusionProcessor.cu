#include "hip/hip_runtime.h"
#include "KinectFusionProcessor.h"
#include "TsdfVolume.h"
//#include "WarpField.h"
#include "device_utils.h"
#include <hip/hip_vector_types.h>
#include ""
#include "GpuMesh.h"
namespace dfusion
{
	__device__ __forceinline__ int sign(float a)
	{
		return (a > 0) - (a < 0);
	}



	texture<depthtype, hipTextureType2D, hipReadModeElementType> g_depth_tex;
	texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> g_color_tex;


#if 1
	struct Fusioner
	{
		PtrStepSz<depthtype> depth;

		hipSurfaceObject_t volumeTex;
		int3 volume_resolution;
		float3 origion;
		float nodeRadius;
		float voxel_size;
		float tranc_dist;
		float max_weight;
		Intr intr;
		float inv_dw_for_fusion2;
		float marchingCube_weightThre;

		Tbx::Mat3 Rv2c;
		Tbx::Point3 tv2c;

		template<int maxK>
		__device__ __forceinline__ void fusion(int x, int y, int z)
		{
			TsdfData rawTsdf = read_tsdf_surface(volumeTex, x, y, z);
			float2 tsdf_weight_prev = unpack_tsdf(rawTsdf);
			float fusion_weight = 0;
			bool suc = true;

			if (!suc)
				return;
			fusion_weight = 1;

			float3 cxyz = convert(Rv2c*(Tbx::Point3(x*voxel_size + origion.x,
				y*voxel_size + origion.y, z*voxel_size + origion.z)) + tv2c);

			float3 uvd = intr.xyz2uvd(cxyz);
			int2 coo = make_int2(__float2int_rn(uvd.x), __float2int_rn(uvd.y));

			if (uvd.x >= 0 && uvd.x < depth.cols && uvd.y >= 0 && uvd.y < depth.rows)
			{
				float depthVal = tex2D(g_depth_tex, coo.x, coo.y)*0.001f;
				float3 dxyz = intr.uvd2xyz(make_float3(coo.x, coo.y, depthVal));
				float sdf = cxyz.z - dxyz.z;

				if (depthVal > KINECT_NEAREST_METER && sdf >= -tranc_dist)
				{
					float tsdf = min(1.0f, sdf / tranc_dist);
					float tsdf_new = (tsdf_weight_prev.x * tsdf_weight_prev.y + fusion_weight * tsdf)
						/ (tsdf_weight_prev.y + fusion_weight);
					float weight_new = min(tsdf_weight_prev.y + fusion_weight, max_weight);
					float4 color = make_float4(0, 0, 0, 0);
					
					write_tsdf_surface(volumeTex, pack_tsdf(tsdf_new, weight_new,
						color), x, y, z);
				}
			}
		}
	};

#endif
	template<int maxK>
	__global__ void tsdf23(Fusioner fs)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		int z = threadIdx.z + blockIdx.z * blockDim.z;

		if (x >= fs.volume_resolution.x || y >= fs.volume_resolution.y || z >= fs.volume_resolution.z)
			return;

		fs.fusion<maxK>(x, y, z);
	}// __global__

	void KinectFusionProcessor::fusion()
	{
		dim3 block(32, 8, 2);
		dim3 grid(divUp(m_volume->getResolution().x, block.x),
			divUp(m_volume->getResolution().y, block.y),
			divUp(m_volume->getResolution().z, block.z));

		// bind src to texture
		g_depth_tex.filterMode = hipFilterModePoint;
		size_t offset;
		hipChannelFormatDesc desc = hipCreateChannelDesc<depthtype>();
		hipBindTexture2D(&offset, &g_depth_tex, m_depth_input.ptr(), &desc,
			m_depth_input.cols(), m_depth_input.rows(), m_depth_input.step());
		assert(offset == 0);
#ifdef ENABLE_COLOR_FUSION
		g_color_tex.filterMode = hipFilterModePoint;
		desc = hipCreateChannelDesc<uchar4>();
		hipBindTexture2D(&offset, &g_color_tex, m_color_input.ptr(), &desc,
			m_color_input.cols(), m_color_input.rows(), m_color_input.step());
		assert(offset == 0);
#endif

		Fusioner fs;
		fs.depth = m_depth_input;
		fs.volumeTex = m_volume->getSurface();
		fs.volume_resolution = m_volume->getResolution();
		fs.origion = m_volume->getOrigion();
		fs.nodeRadius = m_param.warp_radius_search_epsilon;
		fs.voxel_size = m_volume->getVoxelSize();
		fs.tranc_dist = m_volume->getTsdfTruncDist();
		fs.max_weight = m_param.fusion_max_weight;
		fs.intr = m_kinect_intr;
		fs.inv_dw_for_fusion2 = 1.f / (m_param.warp_param_dw_for_fusion*m_param.warp_param_dw_for_fusion);
		fs.marchingCube_weightThre = m_param.marchingCube_min_valied_weight;

		
		Tbx::Transfo tr = rigid_transform;
		fs.Rv2c = tr.get_mat3();
		fs.tv2c = Tbx::Point3(tr.get_translation());

		tsdf23<0> << <grid, block >> >(fs);

		hipUnbindTexture(&g_depth_tex);
#ifdef ENABLE_COLOR_FUSION
		hipUnbindTexture(&g_color_tex);
#endif

		cudaSafeCall(hipGetLastError(), "KinectFusionProcessor::fusion()");
	}

#pragma region --min-filter

	const static int BLOCK_DIM_X = 32;
	const static int BLOCK_DIM_Y = 16;
	const static int MAX_FILTER_RADIUS = 16;
	const static int X_HALO_STEPS = (MAX_FILTER_RADIUS + BLOCK_DIM_X - 1) / BLOCK_DIM_X;
	const static int Y_HALO_STEPS = (MAX_FILTER_RADIUS + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y;
	const static int X_PATCH_PER_BLOCK = 4;
	const static int Y_PATCH_PER_BLOCK = 4;

	template<int Radius>
	__global__ void erose_filter_row(uchar4*  __restrict__ dst,
		const uchar4*  __restrict__ src, int nX, int nY, int pitch)
	{
		// Data cache: threadIdx.x , threadIdx.y
		enum { SMEM_X_LEN = (X_PATCH_PER_BLOCK + 2 * X_HALO_STEPS) * BLOCK_DIM_X };
		enum { SMEM_Y_LEN = BLOCK_DIM_Y };
		__shared__ uchar4 smem[SMEM_Y_LEN][SMEM_X_LEN];

		const int baseX = (blockIdx.x * X_PATCH_PER_BLOCK - X_HALO_STEPS) * BLOCK_DIM_X + threadIdx.x;
		const int baseY = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;
		if (baseY >= nY)
			return;

		src += baseY * pitch + baseX;
		dst += baseY * pitch + baseX;

		//Load main data and right halo
#pragma unroll
		for (int patchId = 0; patchId < X_HALO_STEPS * 2 + X_PATCH_PER_BLOCK; patchId++)
		{
			const int pbx = patchId * BLOCK_DIM_X;
			smem[threadIdx.y][threadIdx.x + pbx] =
				(pbx + baseX < nX && pbx + baseX >= 0) ? src[pbx] : make_uchar4(255, 255, 255, 255);
		}

		//Compute and store results
		__syncthreads();
#pragma unroll
		for (int patchId = X_HALO_STEPS; patchId < X_HALO_STEPS + X_PATCH_PER_BLOCK; patchId++)
		{
			const int pbx = patchId * BLOCK_DIM_X;
			if (baseX + pbx < nX)
			{
				uchar4 s = smem[threadIdx.y][threadIdx.x + pbx];
#pragma unroll
				for (int j = -Radius; j <= Radius; j++)
				{
					if (smem[threadIdx.y][threadIdx.x + pbx + j].x == 0
						&& smem[threadIdx.y][threadIdx.x + pbx + j].y == 0
						&& smem[threadIdx.y][threadIdx.x + pbx + j].z == 0)
						s = make_uchar4(0, 0, 0, 0);
				}
				dst[pbx] = s;
			}
		}
	}

	template<int Radius>
	__global__ void erose_filter_col(uchar4*  __restrict__ dst,
		const uchar4*  __restrict__ src, int nX, int nY, int pitch)
	{
		// Data cache: threadIdx.x , threadIdx.y
		enum { SMEM_X_LEN = BLOCK_DIM_X };
		enum { SMEM_Y_LEN = (Y_PATCH_PER_BLOCK + 2 * Y_HALO_STEPS) * BLOCK_DIM_Y };
		__shared__ uchar4 smem[SMEM_Y_LEN][SMEM_X_LEN];

		const int baseX = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
		const int baseY = (blockIdx.y * Y_PATCH_PER_BLOCK - Y_HALO_STEPS) * BLOCK_DIM_Y + threadIdx.y;
		if (baseX >= nX)
			return;

		src += baseY * pitch + baseX;
		dst += baseY * pitch + baseX;

		//Load main data and lower halo
#pragma unroll
		for (int patchId = 0; patchId < Y_HALO_STEPS * 2 + Y_PATCH_PER_BLOCK; patchId++)
		{
			const int pby = patchId * BLOCK_DIM_Y;
			smem[threadIdx.y + pby][threadIdx.x] =
				(pby + baseY < nY && pby + baseY >= 0) ? src[pby * pitch] : make_uchar4(255, 255, 255, 255);
		}

		//Compute and store results
		__syncthreads();
#pragma unroll
		for (int patchId = Y_HALO_STEPS; patchId < Y_HALO_STEPS + Y_PATCH_PER_BLOCK; patchId++)
		{
			const int pby = patchId * BLOCK_DIM_Y;
			if (baseY + pby < nY)
			{
				uchar4 s = smem[threadIdx.y + pby][threadIdx.x];
#pragma unroll
				for (int j = -Radius; j <= Radius; j++)
				{
					if (smem[threadIdx.y + pby + j][threadIdx.x].x == 0
						&& smem[threadIdx.y + pby + j][threadIdx.x].y == 0
						&& smem[threadIdx.y + pby + j][threadIdx.x].z == 0)
						s = make_uchar4(0, 0, 0, 0);
				}
				dst[pby * pitch] = s;
			}
		}
	}


	template<int Radius>
	static void erose_filter_row_caller(uchar4* dst, const uchar4* src,
		int nX, int nY, int pitch)
	{
		dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
		dim3 grid(divUp(nX, block.x*X_PATCH_PER_BLOCK), divUp(nY, block.y), 1);

		erose_filter_row<Radius> << <grid, block >> >(dst, src, nX, nY, pitch);
	}

	template<int Radius>
	static void erose_filter_col_caller(uchar4* dst, const uchar4* src,
		int nX, int nY, int pitch)
	{
		dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
		dim3 grid(divUp(nX, block.x), divUp(nY, block.y*Y_PATCH_PER_BLOCK), 1);

		erose_filter_col<Radius> << <grid, block >> >(dst, src, nX, nY, pitch);
	}

	void erose_filter(uchar4* dst_d, const uchar4* src_d,
		int nX, int nY, int pitch,
		int radius, int dim)
	{
		if (src_d == dst_d)
			throw std::exception("min_filter: src and dst cannot be the same memory!");
		if (radius <= 0 || radius >= MAX_FILTER_RADIUS)
			throw std::exception("min_filter: error, non supported kernel size!");
		if (dim > 2 || dim < 0)
			throw std::exception("min_filter: illegal input dim");

		typedef void(*row_caller_t)(uchar4* dst, const uchar4* src,
			int nX, int nY, int pitch);
		typedef void(*col_caller_t)(uchar4* dst, const uchar4* src,
			int nX, int nY, int pitch);
		static const row_caller_t row_callers[MAX_FILTER_RADIUS] =
		{
			0, erose_filter_row_caller<1>, erose_filter_row_caller<2>, erose_filter_row_caller<3>,
			erose_filter_row_caller<4>, erose_filter_row_caller<5>, erose_filter_row_caller<6>,
			erose_filter_row_caller<7>, erose_filter_row_caller<8>, erose_filter_row_caller<9>,
			erose_filter_row_caller<10>, erose_filter_row_caller<11>, erose_filter_row_caller<12>,
			erose_filter_row_caller<13>, erose_filter_row_caller<14>, erose_filter_row_caller<15>,
		};
		static const col_caller_t col_callers[MAX_FILTER_RADIUS] =
		{
			0, erose_filter_col_caller<1>, erose_filter_col_caller<2>, erose_filter_col_caller<3>,
			erose_filter_col_caller<4>, erose_filter_col_caller<5>, erose_filter_col_caller<6>,
			erose_filter_col_caller<7>, erose_filter_col_caller<8>, erose_filter_col_caller<9>,
			erose_filter_col_caller<10>, erose_filter_col_caller<11>, erose_filter_col_caller<12>,
			erose_filter_col_caller<13>, erose_filter_col_caller<14>, erose_filter_col_caller<15>,
		};

		if (dim == 0)
		{
			row_callers[radius](dst_d, src_d, nX, nY, pitch);
		}
		if (dim == 1)
		{
			col_callers[radius](dst_d, src_d, nX, nY, pitch);
		}
		cudaSafeCall(hipGetLastError(), "erose_filter");
	}
#pragma endregion

	void KinectFusionProcessor::eroseColor(const ColorMap& src, ColorMap& dst, int nRadius)
	{
		m_color_tmp.create(src.rows(), src.cols());
		dst.create(src.rows(), src.cols());

		erose_filter((uchar4*)m_color_tmp.ptr(), (const uchar4*)src.ptr(), src.cols(),
			src.rows(), src.step() / sizeof(uchar4), nRadius, 0);
		erose_filter((uchar4*)dst.ptr(), (const uchar4*)m_color_tmp.ptr(), src.cols(),
			src.rows(), src.step() / sizeof(uchar4), nRadius, 0);
	}
	struct MeshTrans
	{
		const GpuMesh::PointType* vsrc;
		const GpuMesh::PointType* nsrc;
		const GpuMesh::PointType* csrc;

		GpuMesh::PointType* vdst;
		GpuMesh::PointType* ndst;
		GpuMesh::PointType* cdst;
		int num;

		Tbx::Mat3 R;
		float3 t;

		//float3 origion;
		//float invVoxelSize;

		__device__ __forceinline__ void operator()(int tid)
		{
			float3 p = GpuMesh::from_point(vsrc[tid]);
			float3 n = GpuMesh::from_point(nsrc[tid]);

			Tbx::Point3 p_tbx = Tbx::Point3(convert(p));
			Tbx::Vec3 n_tbx = convert(n);
			vdst[tid] = GpuMesh::to_point(convert(R*p_tbx) + t);
			ndst[tid] = GpuMesh::to_point(convert(R*n_tbx));
			cdst[tid] = csrc[tid];
		}
	};

	__global__ void trans_mesh_kernel(MeshTrans mesh_trans)
	{
		unsigned int i = blockIdx.x * (blockDim.x << 3) + threadIdx.x;

#pragma unroll
		for (int k = 0; k < 8; k++)
		{
			if (i < mesh_trans.num)
			{
				mesh_trans(i);
			}
			i += blockDim.x;
		}
	}
	void KinectFusionProcessor::transGpuMesh(GpuMesh& src, GpuMesh& dst, const Tbx::Transfo &tr)
	{
	


		if (src.num() == 0)
			return;

		dst.create(src.num());
		//printf("src size = %d\n", src.num());
		src.lockVertsNormals();
		dst.lockVertsNormals();

		MeshTrans mesh_trans;
		mesh_trans.t = convert(tr.get_translation());
		mesh_trans.R = tr.get_mat3();// Tbx::Quat_cu(m_rigidTransform);
	
		mesh_trans.vsrc = src.verts();
		mesh_trans.nsrc = src.normals();
		mesh_trans.csrc = src.colors();
		mesh_trans.vdst = dst.verts();
		mesh_trans.ndst = dst.normals();
		mesh_trans.cdst = dst.colors();
		mesh_trans.num = src.num();
		

		dim3 block(512);
		dim3 grid(1, 1, 1);
		grid.x = divUp(dst.num(), block.x << 3);
		trans_mesh_kernel << <grid, block >> >(mesh_trans);
		cudaSafeCall(hipGetLastError(), "trans mesh");

		dst.unlockVertsNormals();
		src.unlockVertsNormals();
	}
}
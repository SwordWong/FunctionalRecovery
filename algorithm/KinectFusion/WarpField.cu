#include "hip/hip_runtime.h"
#include "WarpField.h"
#include "GpuMesh.h"
#include "device_utils.h"
#include "TsdfVolume.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\ModerGpuWrapper.h"
#include "GpuKdTree.h"
#include <set>
#include <algorithm>
#include <queue>
namespace dfusion
{
#pragma region --warpmesh

	struct MeshWarper
	{
		const GpuMesh::PointType* vsrc;
		const GpuMesh::PointType* nsrc;
		const GpuMesh::PointType* csrc;
		hipTextureObject_t knnTex;
		hipTextureObject_t nodesDqVwTex;
		GpuMesh::PointType* vdst;
		GpuMesh::PointType* ndst;
		GpuMesh::PointType* cdst;
		int num;

		Tbx::Mat3 R;
		float3 t;

		float3 origion;
		float invVoxelSize;

		__device__ __forceinline__ void operator()(int tid)
		{
			float3 p = GpuMesh::from_point(vsrc[tid]);
			float3 n = GpuMesh::from_point(nsrc[tid]);

			Tbx::Dual_quat_cu dq_blend = WarpField::calc_dual_quat_blend_on_p(knnTex,
				nodesDqVwTex, p, origion, invVoxelSize);

			Tbx::Point3 dq_p = dq_blend.transform(Tbx::Point3(convert(p)));
			Tbx::Vec3 dq_n = dq_blend.rotate(convert(n));
			
			//vdst[tid] = GpuMesh::to_point(convert(R.rotate(dq_p)) + t);
			//ndst[tid] = GpuMesh::to_point(convert(R.rotate(dq_n)));
			vdst[tid] = GpuMesh::to_point(convert(R*dq_p) + t);
			ndst[tid] = GpuMesh::to_point(convert(R*dq_n));
			cdst[tid] = csrc[tid];

		}
	};

	__global__ void warp_mesh_kernel(MeshWarper warper)
	{
		unsigned int i = blockIdx.x * (blockDim.x << 3) + threadIdx.x;

#pragma unroll
		for (int k = 0; k < 8; k++)
		{
			if (i < warper.num)
			{
				warper(i);
			}
			i += blockDim.x;
		}
	}

	struct MapWarper
	{
		PtrStep<float4> vsrc;
		PtrStep<float4> nsrc;
		hipTextureObject_t knnTex;
		hipTextureObject_t nodesDqVwTex;
		PtrStep<float4> vdst;
		PtrStep<float4> ndst;
		int w;
		int h;

		Tbx::Mat3 R;
		float3 t;

		float3 origion;
		float invVoxelSize;

		__device__ __forceinline__ void operator()(int x, int y)
		{
			float3 p = GpuMesh::from_point(vsrc(y,x));
			float3 n = GpuMesh::from_point(nsrc(y,x));

			Tbx::Dual_quat_cu dq_blend = WarpField::calc_dual_quat_blend_on_p(knnTex,
				nodesDqVwTex, p, origion, invVoxelSize);

			Tbx::Point3 dq_p = dq_blend.transform(Tbx::Point3(convert(p)));
			Tbx::Vec3 dq_n = dq_blend.rotate(convert(n));

			//vdst(y, x) = GpuMesh::to_point(convert(R.rotate(dq_p)) + t);
			//ndst(y, x) = GpuMesh::to_point(convert(R.rotate(dq_n)));
			vdst(y, x) = GpuMesh::to_point(convert(R*dq_p) + t);
			ndst(y, x) = GpuMesh::to_point(convert(R*dq_n));
		}
	};

	__global__ void warp_map_kernel(MapWarper warper)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < warper.w && y < warper.h)
			warper(x, y);
	}

	void WarpField::warp(GpuMesh& src, GpuMesh& dst)
	{
		if (src.num() == 0)
			return;

		dst.create(src.num());

		src.lockVertsNormals();
		dst.lockVertsNormals();

		MeshWarper warper;
		warper.t = convert(m_rigidTransform.get_translation());
		warper.R = m_rigidTransform.get_mat3();// Tbx::Quat_cu(m_rigidTransform);
		warper.knnTex = getKnnFieldTexture();
		warper.nodesDqVwTex = getNodesDqVwTexture();
		warper.vsrc = src.verts();
		warper.nsrc = src.normals();
		warper.csrc = src.colors();
		warper.vdst = dst.verts();
		warper.ndst = dst.normals();
		warper.cdst = dst.colors();
		warper.num = src.num();
		/*warper.origion = m_volume->getOrigion();
		warper.invVoxelSize = 1.f / m_volume->getVoxelSize();*/
		warper.origion = warp_volume_param.warp_volume_original;
		warper.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;

		dim3 block(512);
		dim3 grid(1, 1, 1);
		grid.x = divUp(dst.num(), block.x << 3);
		warp_mesh_kernel << <grid, block >> >(warper);
		cudaSafeCall(hipGetLastError(), "warp mesh");

		dst.unlockVertsNormals();
		src.unlockVertsNormals();
	}

	void WarpField::warp(const MapArr& srcVmap, const MapArr& srcNmap,
		MapArr& dstVmap, MapArr& dstNmap)
	{
		const int w = srcVmap.cols();
		const int h = srcNmap.rows();

		dstVmap.create(h, w);
		dstNmap.create(h, w);

		MapWarper warper;
		warper.t = convert(m_rigidTransform.get_translation());
		warper.R = m_rigidTransform.get_mat3();// Tbx::Quat_cu(m_rigidTransform);
		warper.knnTex = getKnnFieldTexture();
		warper.nodesDqVwTex = getNodesDqVwTexture();
		warper.vsrc = srcVmap;
		warper.nsrc = srcNmap;
		warper.vdst = dstVmap;
		warper.ndst = dstNmap;
		warper.w = w;
		warper.h = h;
		/*warper.origion = m_volume->getOrigion();
		warper.invVoxelSize = 1.f / m_volume->getVoxelSize();*/
		warper.origion = warp_volume_param.warp_volume_original;
		warper.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;

		dim3 block(32, 8);
		dim3 grid(divUp(w, block.x), divUp(h, block.y), 1);
		warp_map_kernel << <grid, block >> >(warper);
		cudaSafeCall(hipGetLastError(), "warp map");
	}
	void WarpField::warp(ObjMesh & src, ObjMesh & dst)
	{
		GpuMesh srcGpu, dstGpu;
		srcGpu.formObjMesh_nonDup(src);
		//dstGpu.create(src.vertex_list.size());

		warp(srcGpu, dstGpu);

		dstGpu.toObjMesh_nonDup(dst);

		srcGpu.release();
		dstGpu.release();
	}
#pragma endregion

#pragma region --init knn field
	__global__ void initKnnFieldKernel(hipSurfaceObject_t knnSurf, int3 resolution)
	{
		int ix = blockDim.x*blockIdx.x + threadIdx.x;
		int iy = blockDim.y*blockIdx.y + threadIdx.y;
		int iz = blockDim.z*blockIdx.z + threadIdx.z;

		if (ix < resolution.x && iy < resolution.y && iz < resolution.z)
			write_knn(make_knn(WarpField::MaxNodeNum), knnSurf, ix, iy, iz);
	}

	__global__ void initKnnFieldKernel1(KnnIdx* knnPtr, int n)
	{
		int ix = blockDim.x*blockIdx.x + threadIdx.x;

		if (ix < n)
			knnPtr[ix] = make_knn(WarpField::MaxNodeNum);
	}

	void WarpField::initKnnField()
	{
		int3 res = warp_volume_param.warp_volume_res;
		dim3 block(32, 8, 2);
		dim3 grid(divUp(res.x, block.x),
			divUp(res.y, block.y),
			divUp(res.z, block.z));

		hipSurfaceObject_t surf = getKnnFieldSurface();
		initKnnFieldKernel << <grid, block >> >(surf, res);
		cudaSafeCall(hipGetLastError(), "initKnnFieldKernel");

		dim3 block1(256);
		dim3 grid1(divUp(m_nodesGraph.size(), block1.x));
		initKnnFieldKernel1 << <grid, block >> >(m_nodesGraph.ptr(), m_nodesGraph.size());
		cudaSafeCall(hipGetLastError(), "initKnnFieldKernel1");
	}
#pragma endregion

#pragma region --update nodes
	__device__ int newPoints_global_count = 0;
	__device__ int newPoints_output_count;
	__device__ unsigned int newPoints_blocks_done = 0;
	struct NewPointsCounter
	{
		enum
		{
			CTA_SIZE = 256,
			WARPS_COUNT = CTA_SIZE / Warp::WARP_SIZE
		};

		mutable int* out_keys;
		mutable float4* out_points;
		GpuMesh::PointType* input_points;
		hipTextureObject_t knnTex;
		hipTextureObject_t nodesDqVwTex;
		float4* nodesDqVw;

		int n;
		int step;
		float3 origion;
		int numNodes;
		float inv_search_radius_sqr;

		// for volume index
		float vol_invVoxelSize;
		int3 vol_res;

		// for key generation
		float key_invGridSize;
		int3 key_gridRes;

		__device__ __forceinline__ void operator () () const
		{
			int tid = threadIdx.x + blockIdx.x * CTA_SIZE;

			if (__all(tid >= n))
				return;

			int warp_id = Warp::id();
			int lane_id = Warp::laneId();
			volatile __shared__ int warps_buffer[WARPS_COUNT];

			int flag = 0;
			int key = 0;
			float4 p4;
			if (tid < n)
			{
				float3 p = GpuMesh::from_point(input_points[tid*step]);
				p4 = GpuMesh::to_point(p, 1.f);

				// generating key
				float3 p1 = (p - origion)*key_invGridSize;
				int x = int(p1.x);
				int y = int(p1.y);
				int z = int(p1.z);

				key = (z*key_gridRes.y + y)*key_gridRes.x + x;

				// identify voxel
				p1 = (p - origion)*vol_invVoxelSize;
				x = int(p1.x);
				y = int(p1.y);
				z = int(p1.z);

				// assert knnIdx sorted, thus the 1st should be the nearest
				KnnIdx knnIdx = read_knn_tex(knnTex, x, y, z);

				if (knn_k(knnIdx, 0) < numNodes)
				{
					float4 nearestVw = make_float4(0, 0, 0, 1);
					tex1Dfetch(&nearestVw, nodesDqVwTex, knn_k(knnIdx, 0) * 3 + 2); // [q0-q1-vw] memory stored

					float3 nearestV = make_float3(nearestVw.x, nearestVw.y, nearestVw.z);

					// DIFFERENT from the paper ldp:
					// here we insert a node if the point is outside the search radius, 
					//  but NOT 1/dw
					// note .w store 1/radius
					float dif = dot(nearestV - p, nearestV - p) * inv_search_radius_sqr;
					flag = (dif > 1.f);
				}
				else
					flag = 1.f;
			}

			int total = __popc(__ballot(flag>0));

			if (total)
			{
				if (lane_id == 0)
				{
					int old = atomicAdd(&newPoints_global_count, total);
					warps_buffer[warp_id] = old;
				}

				int old_global_voxels_count = warps_buffer[warp_id];
				int offs = Warp::binaryExclScan(__ballot(flag>0));
				if (old_global_voxels_count + offs < n && flag)
				{
					out_keys[old_global_voxels_count + offs] = key;
					out_points[old_global_voxels_count + offs] = p4;
				}
			}// end if total

			if (Block::flattenedThreadId() == 0)
			{
				unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
				unsigned int value = atomicInc(&newPoints_blocks_done, total_blocks);

				//last block
				if (value == total_blocks - 1)
				{
					newPoints_output_count = newPoints_global_count;
					newPoints_blocks_done = 0;
					newPoints_global_count = 0;
				}
			}
		} /* operator () */
	};

	__global__ void get_newPoints_kernel(NewPointsCounter counter)
	{
		counter();
	}

	__global__ void pointToKey_kernel(
		const GpuMesh::PointType* points,
		int* key, float4* copypoints, int n, int step,
		float invGridSize, float3 origion, int3 gridRes)
	{
		unsigned int blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
		unsigned int threadId = __mul24(blockId, blockDim.x << 3) + threadIdx.x;

#pragma unroll
		for (int k = 0; k < 8; k++, threadId += blockDim.x)
		{
			if (threadId < n)
			{
				float3 p = GpuMesh::from_point(points[threadId*step]);
				float3 p1 = (p- origion)*invGridSize;
				int x = int(p1.x);
				int y = int(p1.y);
				int z = int(p1.z);
				key[threadId] = (z*gridRes.y + y)*gridRes.x + x;
				copypoints[threadId] = GpuMesh::to_point(p, 1.f);
			}
		}
	}

	__device__ int validVoxel_global_count = 0;
	__device__ int validVoxel_output_count;
	__device__ unsigned int validVoxel_blocks_done = 0;
	struct ValidVoxelCounter
	{
		enum
		{
			CTA_SIZE = 256,
			WARPS_COUNT = CTA_SIZE / Warp::WARP_SIZE
		};

		mutable int* key_sorted;
		mutable int* counts;
		const float4* points_scaned;
		float weight_thre;
		int n;

		__device__ __forceinline__ void operator () () const
		{
			int tid = threadIdx.x + blockIdx.x * CTA_SIZE;

			if (__all(tid >= n))
				return;

			int warp_id = Warp::id();
			int lane_id = Warp::laneId();
			volatile __shared__ int warps_buffer[WARPS_COUNT];

			int flag = 0;
			if (tid < n)
				flag = (points_scaned[tid].w > weight_thre) &&
				(key_sorted[tid] != key_sorted[tid + 1] || tid == n - 1);
			int total = __popc(__ballot(flag>0));

			if (total)
			{
				if (lane_id == 0)
				{
					int old = atomicAdd(&validVoxel_global_count, total);
					warps_buffer[warp_id] = old;
				}

				int old_global_voxels_count = warps_buffer[warp_id];
				int offs = Warp::binaryExclScan(__ballot(flag>0));
				if (old_global_voxels_count + offs < n && flag)
					counts[old_global_voxels_count + offs] = tid;
			}// end if total

			if (Block::flattenedThreadId() == 0)
			{
				unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
				unsigned int value = atomicInc(&validVoxel_blocks_done, total_blocks);

				//last block
				if (value == total_blocks - 1)
				{
					validVoxel_output_count = validVoxel_global_count;
					validVoxel_blocks_done = 0;
					validVoxel_global_count = 0;
				}
			}
		} /* operator () */
	};

	__global__ void get_validVoxel_kernel(ValidVoxelCounter counter)
	{
		counter();
	}

	struct NodesWriter
	{
		const float4* points_not_compact;
		const int* index;
		float4* nodesDqVw;
		float inv_weight_radius;
		int num;

		hipTextureObject_t knnTex;
		hipTextureObject_t nodesDqVwTex;
		float3 origion;
		float invVoxelSize;

		__device__ __forceinline__ void operator()(int threadId)
		{
			int idx = index[threadId];
			float4 p = points_not_compact[idx];
			float inv_w = 1.f / p.w;
			p.x *= inv_w;
			p.y *= inv_w;
			p.z *= inv_w;
			p.w = inv_weight_radius;
			nodesDqVw[threadId * 3 + 2] = p;

			Tbx::Dual_quat_cu dq_blend = WarpField::calc_dual_quat_blend_on_p(knnTex,
				nodesDqVwTex, make_float3(p.x, p.y, p.z), origion, invVoxelSize);

			unpack_dual_quat(dq_blend, nodesDqVw[threadId * 3], nodesDqVw[threadId * 3 + 1]);
		}

		__device__ __forceinline__ void update_nodes_dq_assume_compact_nodes(int threadId)
		{
			float4 p = nodesDqVw[threadId * 3 + 2];
			Tbx::Dual_quat_cu dq_blend = WarpField::calc_dual_quat_blend_on_p(knnTex,
				nodesDqVwTex, make_float3(p.x, p.y, p.z), origion, invVoxelSize);
			unpack_dual_quat(dq_blend, nodesDqVw[threadId * 3], nodesDqVw[threadId * 3 + 1]);
		}
	};

	__global__ void write_nodes_kernel(NodesWriter nw)
	{
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadId < nw.num)
		{
			nw(threadId);
		}
	}

	__global__ void update_nodes_dq_assume_compact_nodes_kernel(NodesWriter nw)
	{
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadId < nw.num)
		{
			nw.update_nodes_dq_assume_compact_nodes(threadId);
		}
	}

	void WarpField::insertNewNodes(GpuMesh& src)
	{
		// make a larger buffer to prevent allocation each time
		int step = m_param.warp_point_step_before_update_node;
		int num_points = src.num() / step;

		if (num_points == 0)
			return;

		if (num_points > m_current_point_buffer_size)
		{
			m_current_point_buffer_size = num_points * 1.5;
			m_meshPointsSorted.create(m_current_point_buffer_size);
			m_meshPointsKey.create(m_current_point_buffer_size);
			m_meshPointsFlags.create(m_current_point_buffer_size);
			m_tmpBuffer.create(m_current_point_buffer_size);

			hipMemset(m_meshPointsSorted.ptr(), 0, m_meshPointsSorted.size()*m_meshPointsSorted.elem_size);
			hipMemset(m_meshPointsKey.ptr(), 0, m_meshPointsKey.size()*m_meshPointsKey.elem_size);
			hipMemset(m_meshPointsFlags.ptr(), 0, m_meshPointsFlags.size()*m_meshPointsFlags.elem_size);
			hipMemset(m_tmpBuffer.ptr(), 0, m_tmpBuffer.size()*m_tmpBuffer.elem_size);
		}

		// reset symbols
		int zero_mem_symbol = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(newPoints_global_count), &zero_mem_symbol, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(newPoints_blocks_done), &zero_mem_symbol, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(validVoxel_global_count), &zero_mem_symbol, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(validVoxel_blocks_done), &zero_mem_symbol, sizeof(int));
		cudaSafeCall(hipDeviceSynchronize(), "set zero: new point");

		// if 1st in, then collect all points
		if (m_lastNumNodes[0] == 0)
		{
			dim3 block(256);
			dim3 grid(1, 1, 1);
			grid.x = divUp(num_points, block.x << 3);

			// copy to new buffer and generate sort key 
			src.lockVertsNormals();
			pointToKey_kernel << <grid, block >> >(
				src.verts(), m_meshPointsKey.ptr(), m_meshPointsSorted.ptr(),
				num_points, step, 1.f / m_param.warp_radius_search_epsilon,
				warp_volume_param.warp_volume_original, m_nodesGridSize);
			cudaSafeCall(hipGetLastError(), "pointToKey_kernel");
			src.unlockVertsNormals();
			printf("step = %d, invGridSize = %f, origion = (%f,%f,%f), gridRes = (%d,%d,%d)\n",
				step, 1.f / m_param.warp_radius_search_epsilon,
				warp_volume_param.warp_volume_original.x, warp_volume_param.warp_volume_original.y, warp_volume_param.warp_volume_original.z,
				m_nodesGridSize.x, m_nodesGridSize.y, m_nodesGridSize.z);
		}
		// else, collect non-covered points
		else
		{
			src.lockVertsNormals();
			NewPointsCounter counter;
			counter.n = num_points;
			counter.step = step;
			counter.origion = warp_volume_param.warp_volume_original;
			counter.key_gridRes = m_nodesGridSize;
			counter.key_invGridSize = 1.f / m_param.warp_radius_search_epsilon;
			counter.vol_invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;
			counter.vol_res = warp_volume_param.warp_volume_res;
			counter.inv_search_radius_sqr = 1.f / (m_param.warp_radius_search_epsilon * 
				m_param.warp_radius_search_epsilon);
			counter.input_points = src.verts();
			counter.out_points = m_meshPointsSorted.ptr();
			counter.out_keys = m_meshPointsKey.ptr();
			counter.knnTex = getKnnFieldTexture();
			counter.nodesDqVwTex = getNodesDqVwTexture();
			counter.nodesDqVw = getNodesDqVwPtr(0);
			counter.numNodes = m_numNodes[0];

			dim3 block1(NewPointsCounter::CTA_SIZE);
			dim3 grid1(divUp(num_points, block1.x));
			get_newPoints_kernel << <grid1, block1 >> >(counter);
			cudaSafeCall(hipGetLastError(), "get_newPoints_kernel");
			cudaSafeCall(hipDeviceSynchronize(), "get_newPoints_kernel sync");

			cudaSafeCall(hipMemcpyFromSymbol(&num_points, HIP_SYMBOL(newPoints_output_count), 
				sizeof(int)), "get_newPoints_kernel memcpy from symbol");

			src.unlockVertsNormals();
		}// end else

		if (num_points == 0)
			return;

		// sort
		thrust_wrapper::sort_by_key(m_meshPointsKey.ptr(), m_meshPointsSorted.ptr(), num_points);

		// segment scan
		thrust_wrapper::inclusive_scan_by_key(m_meshPointsKey.ptr(), 
			m_meshPointsSorted.ptr(), m_meshPointsSorted.ptr(), num_points);

		// compact
		ValidVoxelCounter counter;
		counter.counts = m_meshPointsFlags.ptr();
		counter.key_sorted = m_meshPointsKey.ptr();
		counter.n = num_points;
		counter.weight_thre = m_param.warp_valid_point_num_each_node;
		counter.points_scaned = m_meshPointsSorted.ptr();
		{
			dim3 block1(ValidVoxelCounter::CTA_SIZE);
			dim3 grid1(divUp(num_points, block1.x));
			get_validVoxel_kernel << <grid1, block1 >> >(counter);
			cudaSafeCall(hipGetLastError(), "get_validVoxel_kernel");
			cudaSafeCall(hipDeviceSynchronize(), "get_validVoxel_kernel sync");
		}

		int num_after_compact = 0;
		cudaSafeCall(hipMemcpyFromSymbol(&num_after_compact, 
			HIP_SYMBOL(validVoxel_output_count), sizeof(int)), "copy voxel count from symbol");
		if (num_after_compact == 0 && m_lastNumNodes[0] == 0)
			num_after_compact = 1; // at least one point needed.
		m_numNodes[0] = min(m_lastNumNodes[0] + num_after_compact, MaxNodeNum);
		if (num_after_compact + m_lastNumNodes[0] > MaxNodeNum)
			printf("warning: too many nodes %d vs %d\n", num_after_compact + m_lastNumNodes[0], MaxNodeNum);

		if (m_numNodes[0] > m_lastNumNodes[0])
		{
			dim3 block(256);
			dim3 grid(1, 1, 1);
			grid.x = divUp(m_numNodes[0] - m_lastNumNodes[0], block.x);

			NodesWriter nw;
			nw.points_not_compact = m_meshPointsSorted.ptr();
			nw.index = m_meshPointsFlags.ptr();
			nw.nodesDqVw = getNodesDqVwPtr(0) + m_lastNumNodes[0] * 3;
			nw.num = m_numNodes[0] - m_lastNumNodes[0];
			nw.inv_weight_radius = 1.f / m_param.warp_param_dw;
			nw.origion = warp_volume_param.warp_volume_original;
			nw.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;
			nw.knnTex = getKnnFieldTexture();
			nw.nodesDqVwTex = getNodesDqVwTexture();

			write_nodes_kernel << <grid, block >> >(nw);
			cudaSafeCall(hipGetLastError(), "write_nodes_kernel");
		}
	}
#pragma endregion

#pragma region --update ann field
	__global__ void seperate_xyz_nodes(const float4* nodesDqVw, 
		float* x, float* y, float* z, int n)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < n)
		{
			float4 dqVw = nodesDqVw[tid * 3 + 2];
			x[tid] = dqVw.x;
			y[tid] = dqVw.y;
			z[tid] = dqVw.z;
		}
	}

	__global__ void collect_aabb_box_kernel(float4* aabb_min, float4* aabb_max,
		const float* x, const float* y, const float* z, int n)
	{
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		if (tid == 0)
		{
			aabb_min[0] = make_float4(x[0], y[0], z[0], 0);
			aabb_max[0] = make_float4(x[n-1], y[n - 1], z[n - 1], 0);
		}
	}

	__global__ void bruteforce_updateKnn_kernel(hipTextureObject_t nodesDqVwTex,
		hipSurfaceObject_t knnSurf, int3 res, int newNodesBegin, int newNodesEnd,
		float3 origion, float voxelSize, int maxK)
	{
		int x = threadIdx.x + blockIdx.x*blockDim.x;
		int y = threadIdx.y + blockIdx.y*blockDim.y;
		int z = threadIdx.z + blockIdx.z*blockDim.z;

		if (x < res.x && y < res.y && z < res.z)
		{
			// compute all 4 dists stored
			KnnIdx knn = read_knn_surf(knnSurf, x, y, z);
			float3 voxelPos = origion + voxelSize*make_float3(x, y, z);
			float oldDists2[KnnK];
			for (int k = 0; k < maxK; k++)
			{
				float4 p;
				tex1Dfetch(&p, nodesDqVwTex, knn_k(knn, k)*3 + 2);
				oldDists2[k] = norm2(make_float3(p.x, p.y, p.z) - voxelPos);
			}
			
			// update new nodes
			for (int iNode = newNodesBegin; iNode < newNodesEnd; iNode++)
			{
				float4 p;
				tex1Dfetch(&p, nodesDqVwTex, iNode * 3 + 2);
				float newDist2 = norm2(make_float3(p.x, p.y, p.z) - voxelPos);

				// we swap the farest nodes out
				// note that the knn is kept sorted
				int swapPos = maxK;
				for (int k = 0; k < maxK; k++)
				{
					if (newDist2 < oldDists2[k])
					{
						swapPos = k;
						break;
					}
				}

				if (swapPos < maxK)
				{
					KnnIdx newKnn = knn;
					knn_k(newKnn, swapPos) = iNode;
					for (int k = swapPos + 1; k < maxK; k++)
						knn_k(newKnn, k) = knn_k(knn, k - 1);
					write_knn(newKnn, knnSurf, x, y, z);
				}
			}// end for iNode
		}
	}

	void WarpField::updateAnnField()
	{
		float3 origion = warp_volume_param.warp_volume_original;
		int3 res = warp_volume_param.warp_volume_res;
		float vsz = warp_volume_param.warp_voxel_size;

		// if 1st frame, then perform whole-volume search, which is slow
		if (m_lastNumNodes[0] == 0)
		{
			m_nodeTree[0]->buildTree(m_nodesQuatTransVw.ptr() + 2, m_numNodes[0], 3);
			hipSurfaceObject_t surf = getKnnFieldSurface();
			m_nodeTree[0]->knnSearchGpu(surf, make_int3(0, 0, 0), res, origion, vsz, 
				m_param.warp_knn_k_eachlevel[0]);
		}
		// else, collect voxels around the new added node and then perform sub-volume searching
		else
		{
			int nNewNodes = m_numNodes[0] - m_lastNumNodes[0];
#if 0
			// 1st step, collect bounding box of new nodes to avoid additional computation
			float* xptr = m_tmpBuffer.ptr() + nNewNodes;
			float* yptr = xptr + nNewNodes;
			float* zptr = yptr + nNewNodes;
			if (nNewNodes)
			{
				dim3 block(32);
				dim3 grid(divUp(nNewNodes, block.x));
				seperate_xyz_nodes << <grid, block >> >(getNodesDqVwPtr(0) + m_lastNumNodes[0] * 3, 
					xptr, yptr, zptr, nNewNodes);
				cudaSafeCall(hipGetLastError(), "seperate_xyz_nodes");
			}

			modergpu_wrapper::mergesort(xptr, nNewNodes);
			modergpu_wrapper::mergesort(yptr, nNewNodes);
			modergpu_wrapper::mergesort(zptr, nNewNodes);

			// bounding box info
			float4 box[2];
			{
				dim3 block(1);
				dim3 grid(1);
				collect_aabb_box_kernel << <grid, block >> >(
					m_meshPointsSorted.ptr(), m_meshPointsSorted.ptr() + 1, xptr, yptr, zptr, nNewNodes);
				cudaSafeCall(hipGetLastError(), "collect_aabb_box_kernel");
				cudaSafeCall(hipMemcpy(box, m_meshPointsSorted.ptr(), 2 * sizeof(float4), 
					hipMemcpyDeviceToHost));
			}

			// convert to volume index
			int3 begin = make_int3((box[0].x - origion.x) / vsz, 
				(box[0].y - origion.y) / vsz, (box[0].z - origion.z) / vsz);
			int3 end = make_int3((box[1].x - origion.x) / vsz + 1,
				(box[1].y - origion.y) / vsz + 1, (box[1].z - origion.z) / vsz + 1);
			int ext = ceil(m_param.warp_param_dw / vsz);
			begin.x = min(res.x - 1, max(0, begin.x - ext));
			begin.y = min(res.y - 1, max(0, begin.y - ext));
			begin.z = min(res.z - 1, max(0, begin.z - ext));
			end.x = max(1, min(res.x, end.x + ext));
			end.y = max(1, min(res.y, end.y + ext));
			end.z = max(1, min(res.z, end.z + ext));

			// perform knn search on the sub volume
			m_nodeTree[0]->buildTree(m_nodesQuatTransVw.ptr() + 2, m_numNodes[0], 3);
			hipSurfaceObject_t surf = bindKnnFieldSurface();
			m_nodeTree[0]->knnSearchGpu(surf, begin, end, origion, vsz, KnnK);
			//m_nodeTree[0]->knnSearchGpu(surf, make_int3(0,0,0), res, origion, vsz, KnnK);
			unBindKnnFieldSurface(surf);
#else
			//tranverse each voxel to update
			if (nNewNodes > 0)
			{
				int3 res = warp_volume_param.warp_volume_res;
				float3 origion = warp_volume_param.warp_volume_original;
				float vsz = warp_volume_param.warp_voxel_size;
				dim3 block(32, 8, 2);
				dim3 grid(divUp(res.x, block.x),
					divUp(res.y, block.y),
					divUp(res.z, block.z));

				hipSurfaceObject_t surf = getKnnFieldSurface();
				hipTextureObject_t tex = getNodesDqVwTexture();
				bruteforce_updateKnn_kernel << <grid, block >> >(
					tex, surf, res, m_lastNumNodes[0], m_numNodes[0], origion, vsz,
					m_param.warp_knn_k_eachlevel[0]);
				cudaSafeCall(hipGetLastError(), "bruteforce_updateKnn_kernel");
			}
#endif
		}
	}
#pragma endregion

#pragma region remove small graph components

	struct sort_int2_less
	{
		bool operator()(const int2& left, const int2& right)const
		{
			return (left.x < right.x) || (left.x == right.x && left.y < right.y);
		}
	};

	__global__ void copy_nodes_kernel(float4* dst, const float4* src, const int* idxMap, int nSrc)
	{
		int iSrc = threadIdx.x + blockIdx.x * blockDim.x;
		if (iSrc < nSrc)
		{
			int iDst = idxMap[iSrc];
			if (iDst >= 0)
			{
				for (int k = 0; k < 3; k++)
					dst[iDst * 3 + k] = src[iSrc * 3 + k];
			}
		}
	}

	void WarpField::remove_small_graph_components()
	{
		// we only perform removal for single-level graph
		if (!m_param.graph_single_level || m_numNodes[0] <= 1
			|| m_param.graph_remove_small_components_ratio >= 1.f
			|| m_numNodes[0] == m_lastNumNodes[0])
			return;

		std::vector<KnnIdx> knnGraph(m_numNodes[0]);
		cudaSafeCall(hipMemcpy(knnGraph.data(), m_nodesGraph.ptr(), m_numNodes[0] * sizeof(KnnIdx),
			hipMemcpyDeviceToHost), "WarpField::remove_small_graph_components, cudaMemcpy1");

		std::vector<int2> edges;
		edges.reserve(knnGraph.size() * KnnK);
		for (int i = 0; i < knnGraph.size(); i++)
		{
			KnnIdx knn = knnGraph[i];
			for (int k = 0; k < KnnK; k++)
			{
				int nb = knn_k(knn, k);
				if (nb < m_numNodes[0])
				{
					edges.push_back(make_int2(i, nb));
					edges.push_back(make_int2(nb, i));
				}
			}// k
		}// i
		std::sort(edges.begin(), edges.end(), sort_int2_less());

		std::vector<int> edgeHeader(m_numNodes[0] + 1, 0);
		for (int i = 1; i < edges.size(); i++)
		{
			if (edges[i].x != edges[i - 1].x)
				edgeHeader[edges[i].x] = i;
		}
		edgeHeader[m_numNodes[0]] = edges.size();

		// find indepedent components
		std::set<int> verts;
		for (int i = 0; i < m_numNodes[0]; i++)
			verts.insert(i);

		std::vector<int> componentsSize;
		std::vector<int> componentsFlag(m_numNodes[0], -1);

		while (!verts.empty())
		{
			componentsSize.push_back(0);
			int& cpSz = componentsSize.back();

			auto set_iter = verts.begin();
			std::queue<int> queue;
			queue.push(*set_iter);
			verts.erase(set_iter);

			while (!queue.empty())
			{
				const int v = queue.front();
				queue.pop();
				cpSz++;
				componentsFlag[v] = componentsSize.size() - 1;

				for (int i = edgeHeader[v]; i < edgeHeader[v + 1]; i++)
				{
					const int v1 = edges[i].y;
					set_iter = verts.find(v1);
					if (set_iter != verts.end())
					{
						queue.push(v1);
						verts.erase(set_iter);
					}
				}// end for i
			}// end while
		}// end while verts

		// if only one components, then nothing to remove
		if (componentsSize.size() <= 1)
			return;

		// find idx that map origional nodes to removed nodes set
		const int thre = std::lroundf(m_param.graph_remove_small_components_ratio * m_numNodes[0]);
		std::set<int> componentsToRemove;
		for (int i = 0; i < componentsSize.size(); i++)
		if (componentsSize[i] < thre)
			componentsToRemove.insert(i);

		if (componentsToRemove.size() == 0)
			return;

		int totalIdx = 0;
		std::vector<int> idxMap(componentsFlag.size());
		for (int i = 0; i < componentsFlag.size(); i++)
		{
			if (componentsToRemove.find(componentsFlag[i]) != componentsToRemove.end())
			{
				idxMap[i] = -1;
				if (i < m_lastNumNodes[0])
				{
					//printf("illegal: %d < %d, current: %d\n", i, m_lastNumNodes[0], m_numNodes[0]);
					//throw std::exception("error in removing small components, last nodes not illegal!");
					idxMap[i] = totalIdx++;
				}
			}
			else
				idxMap[i] = totalIdx++;
		}

		//
		if (m_meshPointsKey.size() < m_numNodes[0])
			m_meshPointsKey.create(m_numNodes[0] * 1.5);
		if (m_meshPointsSorted.size() < m_numNodes[0] * 3)
			m_meshPointsSorted.create(m_numNodes[0] * 3 * 1.5);
		cudaSafeCall(hipMemcpy(m_meshPointsSorted, m_nodesQuatTransVw, m_numNodes[0] * sizeof(float4)* 3,
			hipMemcpyDeviceToDevice), "WarpField::remove_small_graph_components, cudaMemcpy2");
		cudaSafeCall(hipMemcpy(m_meshPointsKey, idxMap.data(), m_numNodes[0] * sizeof(int),
			hipMemcpyHostToDevice), "WarpField::remove_small_graph_components, cudaMemcpy3");
		copy_nodes_kernel << <divUp(m_numNodes[0], 256), 256 >> >(m_nodesQuatTransVw,
			m_meshPointsSorted, m_meshPointsKey, m_numNodes[0]);
		cudaSafeCall(hipGetLastError(), "WarpField::remove_small_graph_components, copy nodes");

		printf("Nodes Removal: %d -> %d, last=%d\n", m_numNodes[0], totalIdx, m_lastNumNodes[0]);
		m_numNodes[0] = totalIdx;

		updateGraph_singleLevel();
	}
#pragma endregion

#pragma region --update graph
	void WarpField::updateGraph(int level)
	{
		if (level == 0)
			throw std::exception("called an invalid level function\n");

		int num_points = m_numNodes[level - 1];

		if (num_points == 0)
		{
			m_numNodes[level] = 0;
			return;
		}

		// re-define structure only if lv0 structure changed===============================
		if (m_lastNumNodes[0] != m_numNodes[0])
		{
			// reset symbols
			int zero_mem_symbol = 0;
			hipMemcpyToSymbol(HIP_SYMBOL(newPoints_global_count), &zero_mem_symbol, sizeof(int));
			hipMemcpyToSymbol(HIP_SYMBOL(newPoints_blocks_done), &zero_mem_symbol, sizeof(int));
			hipMemcpyToSymbol(HIP_SYMBOL(validVoxel_global_count), &zero_mem_symbol, sizeof(int));
			hipMemcpyToSymbol(HIP_SYMBOL(validVoxel_blocks_done), &zero_mem_symbol, sizeof(int));
			cudaSafeCall(hipDeviceSynchronize(), "set zero: new point");

			float radius = m_param.warp_radius_search_epsilon * pow(m_param.warp_radius_search_beta, level);

			{
				dim3 block(32);
				dim3 grid(1, 1, 1);
				grid.x = divUp(num_points, block.x << 3);

				// copy to new buffer and generate sort key
				pointToKey_kernel << <grid, block >> >(
					getNodesDqVwPtr(level - 1) + 2, m_meshPointsKey.ptr(), m_meshPointsSorted.ptr(),
					num_points, 3, 1.f / radius, warp_volume_param.warp_volume_original, m_nodesGridSize);
				cudaSafeCall(hipGetLastError(), "pointToKey_kernel lv");
				
			}

			if (num_points == 0)
				return;

			// sort
			thrust_wrapper::sort_by_key(m_meshPointsKey.ptr(), m_meshPointsSorted.ptr(), num_points);

			// segment scan
			thrust_wrapper::inclusive_scan_by_key(m_meshPointsKey.ptr(),
				m_meshPointsSorted.ptr(), m_meshPointsSorted.ptr(), num_points);

			// compact
			ValidVoxelCounter counter;
			counter.counts = m_meshPointsFlags.ptr();
			counter.key_sorted = m_meshPointsKey.ptr();
			counter.n = num_points;
			counter.weight_thre = 1;
			counter.points_scaned = m_meshPointsSorted.ptr();
			if (num_points)
			{
				dim3 block1(ValidVoxelCounter::CTA_SIZE);
				dim3 grid1(divUp(num_points, block1.x));
				get_validVoxel_kernel << <grid1, block1 >> >(counter);
				cudaSafeCall(hipGetLastError(), "get_validVoxel_kernel lv");
				cudaSafeCall(hipDeviceSynchronize(), "get_validVoxel_kernel lv sync");
			}

			int num_after_compact = 0;
			cudaSafeCall(hipMemcpyFromSymbol(&num_after_compact,
				HIP_SYMBOL(validVoxel_output_count), sizeof(int)), "copy voxel count from symbol");
			m_numNodes[level] = min(num_after_compact, MaxNodeNum);
			if (num_after_compact > MaxNodeNum)
				printf("warning: too many nodes %d vs %d in level\n",
				num_after_compact + m_lastNumNodes[0], MaxNodeNum, level);

			// write level nodes
			if (m_numNodes[level] > 0)
			{
				dim3 block(32);
				dim3 grid(1, 1, 1);
				grid.x = divUp(m_numNodes[level], block.x);

				NodesWriter nw;
				nw.points_not_compact = m_meshPointsSorted.ptr();
				nw.index = m_meshPointsFlags.ptr();
				nw.nodesDqVw = getNodesDqVwPtr(level);
				nw.num = m_numNodes[level];
				nw.inv_weight_radius = 1.f / (m_param.warp_param_dw*pow(m_param.warp_radius_search_beta, level));
				nw.origion = warp_volume_param.warp_volume_original;
				nw.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;
				nw.knnTex = getKnnFieldTexture();
				nw.nodesDqVwTex = getNodesDqVwTexture();

				write_nodes_kernel << <grid, block >> >(nw);
				cudaSafeCall(hipGetLastError(), "write_nodes_kernel");
			}

			// build graph
			if (m_numNodes[level] > 0)
			{
				m_nodeTree[level]->buildTree(getNodesDqVwPtr(level) + 2, m_numNodes[level], 3);

				dim3 block1(256);
				dim3 grid1(divUp(getNumNodesInLevel(level-1)*KnnK, block1.x));
				initKnnFieldKernel1 << <grid1, block1 >> >(getNodesEdgesPtr(level - 1), 
					getNumNodesInLevel(level - 1)*KnnK);
				cudaSafeCall(hipGetLastError(), "initKnnFieldKernel1-1");

				m_nodeTree[level]->knnSearchGpu(getNodesDqVwPtr(level - 1) + 2, 3,
					(KnnIdxType*)getNodesEdgesPtr(level - 1), nullptr, m_param.warp_knn_k_eachlevel[level], 
					getNumNodesInLevel(level - 1), KnnK);
			}
		}// end if (m_lastNumNodes[0] != m_numNodes[0])
		else if (m_numNodes[level])// else we only update the graph quaternions
		{
			dim3 block(32);
			dim3 grid(1, 1, 1);
			grid.x = divUp(m_numNodes[level], block.x);

			NodesWriter nw;
			nw.nodesDqVw = getNodesDqVwPtr(level);
			nw.num = m_numNodes[level];
			nw.inv_weight_radius = 1.f / (m_param.warp_param_dw*pow(m_param.warp_param_dw_lvup_scale, level));
			nw.origion = warp_volume_param.warp_volume_original;
			nw.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;
			nw.knnTex = getKnnFieldTexture();
			nw.nodesDqVwTex = getNodesDqVwTexture();

			update_nodes_dq_assume_compact_nodes_kernel << <grid, block >> >(nw);
			cudaSafeCall(hipGetLastError(), "update_nodes_dq_assume_compact_nodes_kernel");
		}// end else (m_lastNumNodes[0] == m_numNodes[0])
	}

	void WarpField::updateGraph_singleLevel()
	{
		// build graph
		if (m_lastNumNodes[0] != m_numNodes[0])
		{
			m_nodeTree[0]->buildTree(getNodesDqVwPtr(0) + 2, m_numNodes[0], 3);

			dim3 block1(256);
			dim3 grid1(divUp(getNumNodesInLevel(0)*KnnK, block1.x));
			initKnnFieldKernel1 << <grid1, block1 >> >(getNodesEdgesPtr(0),
				getNumNodesInLevel(0)*KnnK);
			cudaSafeCall(hipGetLastError(), "initKnnFieldKernel1-1");

			m_nodeTree[0]->knnSearchGpu(getNodesDqVwPtr(0) + 2, 3,
				(KnnIdxType*)getNodesEdgesPtr(0), nullptr, m_param.warp_knn_k_eachlevel[1],
				getNumNodesInLevel(0), KnnK, m_param.graph_single_level);
		}
		else if (m_numNodes[0])// else we only update the graph quaternions
		{
			dim3 block(32);
			dim3 grid(1, 1, 1);
			grid.x = divUp(m_numNodes[0], block.x);

			NodesWriter nw;
			nw.nodesDqVw = getNodesDqVwPtr(0);
			nw.num = m_numNodes[0];
			nw.inv_weight_radius = 1.f / m_param.warp_param_dw;
			nw.origion = warp_volume_param.warp_volume_original;
			nw.invVoxelSize = 1.f / warp_volume_param.warp_voxel_size;
			nw.knnTex = getKnnFieldTexture();
			nw.nodesDqVwTex = getNodesDqVwTexture();

			update_nodes_dq_assume_compact_nodes_kernel << <grid, block >> >(nw);
			cudaSafeCall(hipGetLastError(), "update_nodes_dq_assume_compact_nodes_kernel");
		}// end else (m_lastNumNodes[0] == m_numNodes[0])
	}
#pragma endregion
	
#pragma region --extract_for_vmap
	struct IdxContainter
	{
		int id[WarpField::GraphLevelNum+1];
		__device__ __host__ int& operator [](int i)
		{
			return id[i];
		}
	};

	__global__ void extract_knn_for_vmap_kernel(PtrStepSz<float4> vmap, PtrStepSz<KnnIdx> vmapKnn,
		float3 origion, float invVoxelSize, hipTextureObject_t knnTex, IdxContainter ic)
	{
		int u = blockIdx.x * blockDim.x + threadIdx.x;
		int v = blockIdx.y * blockDim.y + threadIdx.y;

		if (u < vmap.cols && v < vmap.rows)
		{
			float3 p = GpuMesh::from_point(vmap(v, u));
			KnnIdx knnIdx = make_knn(ic[WarpField::GraphLevelNum]);

			if (!isnan(p.x))
			{
				float3 p1 = (p - origion)*invVoxelSize;
				int x = int(p1.x);
				int y = int(p1.y);
				int z = int(p1.z);
				knnIdx = read_knn_tex(knnTex, x, y, z);
				for (int k = 0; k < KnnK; k++)
				{
					if (knn_k(knnIdx, k) >= WarpField::MaxNodeNum)
						knn_k(knnIdx, k) = ic[WarpField::GraphLevelNum];
				}
			}

			vmapKnn(v, u) = knnIdx;
		}
	}

	void WarpField::extract_knn_for_vmap(const MapArr& vmap, DeviceArray2D<KnnIdx>& vmapKnn)const
	{
		IdxContainter ic;
		ic[0] = 0;
		for (int k = 0; k < GraphLevelNum; k++)
			ic[k + 1] = ic[k] + m_numNodes[k];

		vmapKnn.create(vmap.rows(), vmap.cols());

		dim3 block(32, 8);
		dim3 grid(divUp(vmap.cols(), block.x), divUp(vmap.rows(), block.y));

		hipTextureObject_t knnTex = getKnnFieldTexture();
		extract_knn_for_vmap_kernel << <grid, block >> >(vmap, vmapKnn, warp_volume_param.warp_volume_original,
			1.f / warp_volume_param.warp_voxel_size, knnTex, ic);
		cudaSafeCall(hipGetLastError(), "extract_knn_for_vmap_kernel");
		
	}

	__global__ void extract_nodes_info_kernel(const float4* nodesDqVw, float* twist, float4* vw,
		const KnnIdx* nodesKnnIn, KnnIdx* nodesKnnOut, 
		IdxContainter ic, bool single_graph_level)
	{
		int iout = blockIdx.x * blockDim.x + threadIdx.x;
		if (iout >= ic[WarpField::GraphLevelNum])
			return;

		int level = 0;
		for (int k = 0; k < WarpField::GraphLevelNum; k++)
		if (iout >= ic[k] && iout < ic[k + 1])
		{
			level = k;
			break;
		}

		int iin = level*WarpField::MaxNodeNum + iout - ic[level];

		// write twist
		Tbx::Dual_quat_cu dq = pack_dual_quat(nodesDqVw[iin * 3], nodesDqVw[iin * 3 + 1]);
		Tbx::Vec3 r, t;
		dq.to_twist(r, t);
		twist[iout * 6 + 0] = r.x;
		twist[iout * 6 + 1] = r.y;
		twist[iout * 6 + 2] = r.z;
		twist[iout * 6 + 3] = t.x;
		twist[iout * 6 + 4] = t.y;
		twist[iout * 6 + 5] = t.z;
		vw[iout] = nodesDqVw[iin * 3 + 2];

		// write knn
		KnnIdx kid = nodesKnnIn[iin];
		for (int k = 0; k < KnnK; k++)
		{
			if (!single_graph_level)
				knn_k(kid, k) = (knn_k(kid, k) < ic[level + 1] - ic[level] ? 
					knn_k(kid, k) + ic[level + 1] : ic[WarpField::GraphLevelNum]);
			else
				knn_k(kid, k) = (knn_k(kid, k) < WarpField::MaxNodeNum ?
					knn_k(kid, k) : ic[WarpField::GraphLevelNum]);
		}
		nodesKnnOut[iout] = kid;
	}

	void WarpField::extract_nodes_info(DeviceArray<KnnIdx>& nodesKnn, DeviceArray<float>& twist,
		DeviceArray<float4>& vw)const
	{
		IdxContainter ic;
		ic[0] = 0;
		for (int k = 0; k < GraphLevelNum; k++)
			ic[k + 1] = ic[k] + m_numNodes[k];

		if (ic[GraphLevelNum] == 0)
			return;

		nodesKnn.create(ic[GraphLevelNum]);
		twist.create(ic[GraphLevelNum] * 6);
		vw.create(ic[GraphLevelNum]);

		extract_nodes_info_no_allocation(nodesKnn, twist, vw);
	}

	void WarpField::extract_nodes_info_no_allocation(
		DeviceArray<KnnIdx>& nodesKnn,
		DeviceArray<float>& twist,
		DeviceArray<float4>& vw)const
	{
		IdxContainter ic;
		ic[0] = 0;
		for (int k = 0; k < GraphLevelNum; k++)
			ic[k + 1] = ic[k] + m_numNodes[k];

		if (ic[GraphLevelNum] == 0)
			return;

		dim3 block(256);
		dim3 grid(divUp(ic[GraphLevelNum], block.x));
		
		extract_nodes_info_kernel << <grid, block >> >(getNodesDqVwPtr(0),
			twist.ptr(), vw.ptr(), getNodesEdgesPtr(0), nodesKnn.ptr(), ic,
			m_param.graph_single_level);
		cudaSafeCall(hipGetLastError(), "extract_nodes_info_kernel");
	}

	__global__ void update_nodes_via_twist_kernel(float4* nodesDqVw, const float* twist,
		IdxContainter ic)
	{
		int iout = blockIdx.x * blockDim.x + threadIdx.x;
		if (iout >= ic[WarpField::GraphLevelNum])
			return;

		int level = 0;
		for (int k = 0; k < WarpField::GraphLevelNum; k++)
		if (iout >= ic[k] && iout < ic[k + 1])
		{
			level = k;
			break;
		}

		int iin = level*WarpField::MaxNodeNum + iout - ic[level];

		// write twist
		Tbx::Vec3 r, t;
		r.x = twist[iout * 6 + 0];
		r.y = twist[iout * 6 + 1];
		r.z = twist[iout * 6 + 2];
		t.x = twist[iout * 6 + 3];
		t.y = twist[iout * 6 + 4];
		t.z = twist[iout * 6 + 5];
		Tbx::Dual_quat_cu dq;
		dq.from_twist(r, t);
		unpack_dual_quat(dq, nodesDqVw[iin * 3], nodesDqVw[iin * 3 + 1]);
	}

	void WarpField::update_nodes_via_twist(const DeviceArray<float>& twist)
	{
		IdxContainter ic;
		ic[0] = 0;
		for (int k = 0; k < GraphLevelNum; k++)
			ic[k + 1] = ic[k] + m_numNodes[k];

		if (twist.size() < ic[GraphLevelNum]*6)
			throw std::exception("size not matched in WarpField::update_nodes_via_twist()");

		dim3 block(256);
		dim3 grid(divUp(ic[GraphLevelNum], block.x));

		update_nodes_via_twist_kernel << <grid, block >> >(getNodesDqVwPtr(0),
			twist.ptr(), ic);
		cudaSafeCall(hipGetLastError(), "update_nodes_via_twist");
	}
#pragma endregion


#pragma region --extract_for_verts
	__global__ void extract_knnIdx_for_verts_kernel(
		int n, const GpuMesh::PointType* verts, KnnIdx* knns,
		float3 origion, float invVoxelSize, hipTextureObject_t knnTex, IdxContainter ic)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;

		if (idx < n)
		{
			float3 p = GpuMesh::from_point(verts[idx]);
			KnnIdx knnIdx = make_knn(ic[WarpField::GraphLevelNum]);

			if (!isnan(p.x))
			{
				float3 p1 = (p - origion)*invVoxelSize;
				int x = int(p1.x);
				int y = int(p1.y);
				int z = int(p1.z);
				knnIdx = read_knn_tex(knnTex, x, y, z);
				for (int k = 0; k < KnnK; k++)
				{
					if (knn_k(knnIdx, k) >= WarpField::MaxNodeNum)
						knn_k(knnIdx, k) = ic[WarpField::GraphLevelNum];
				}
			}

			knns[idx] = knnIdx;
		}
	}

	void WarpField::extract_knnIdx_for_verts(GpuMesh& src, DeviceArray<KnnIdx>& knns)
	{
		IdxContainter ic;
		ic[0] = 0;
		for (int k = 0; k < GraphLevelNum; k++)
			ic[k + 1] = ic[k] + m_numNodes[k];

		knns.create(src.num());

		dim3 block(512);
		dim3 grid(divUp(src.num(), block.x));

		src.lockVertsNormals();

		hipTextureObject_t knnTex = getKnnFieldTexture();
		extract_knnIdx_for_verts_kernel << <grid, block >> >(src.num(), src.verts(), knns, warp_volume_param.warp_volume_original,
			1.f / warp_volume_param.warp_voxel_size, knnTex, ic);
		cudaSafeCall(hipGetLastError(), "extract_knnIdx_for_verts_kernel");
		
		src.unlockVertsNormals();
	}
#pragma endregion

#pragma region --getKnnAt

	__global__ void getKnnAtKernel(KnnIdx* data, int3 p, hipTextureObject_t tex)
	{
		data[0] = read_knn_tex(tex, p.x, p.y, p.z);
	}

	KnnIdx WarpField::getKnnAt(float3 volumePos)const
	{
		/*if (m_volume == nullptr)
			throw std::exception("WarpField::getKnnAt(): null pointer");*/
		float3 ori = warp_volume_param.warp_volume_original;
		float vsz = warp_volume_param.warp_voxel_size;
		float3 p = (volumePos - ori) / vsz;
		return getKnnAt(make_int3(p.x, p.y, p.z));
	}
	KnnIdx WarpField::getKnnAt(int3 gridXYZ)const
	{
		/*if (m_volume == nullptr)
			throw std::exception("WarpField::getKnnAt(): null pointer");*/
		int3 res = warp_volume_param.warp_volume_res;
		int x = gridXYZ.x, y = gridXYZ.y, z = gridXYZ.z;
		if (x < 0 || y < 0 || z < 0 || x >= res.x || y >= res.y || z >= res.z)
			return make_knn(MaxNodeNum);
		static DeviceArray<KnnIdx> knn;
		knn.create(1);

		hipTextureObject_t tex = getKnnFieldTexture();
		getKnnAtKernel << <dim3(1), dim3(1) >> >(knn.ptr(), gridXYZ, tex);
		cudaSafeCall(hipGetLastError(), "WarpField::getKnnAtKernel");

		KnnIdx host;
		cudaSafeCall(hipMemcpy(&host, knn.ptr(), sizeof(KnnIdx), hipMemcpyDeviceToHost),
			"WarpField::getKnnAtKernel, post copy");
		return host;
	}
#pragma endregion
}